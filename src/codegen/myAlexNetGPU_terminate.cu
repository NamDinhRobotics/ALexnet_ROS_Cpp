//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: myAlexNetGPU_terminate.cu
//
// GPU Coder version                    : 2.2
// CUDA/C/C++ source code generated on  : 10-Dec-2021 14:26:27
//

// Include Files
#include "myAlexNetGPU_terminate.h"
#include "myAlexNetGPU.h"
#include "myAlexNetGPU_data.h"
#include <cstdio>

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void myAlexNetGPU_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "ERR[%d] %s:%s\n", errCode, hipGetErrorName(errCode),
            hipGetErrorString(errCode));
    exit(errCode);
  }
  myAlexNetGPU_free();
  isInitialized_myAlexNetGPU = false;
}

//
// File trailer for myAlexNetGPU_terminate.cu
//
// [EOF]
//
