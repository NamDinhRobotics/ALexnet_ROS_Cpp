#include "hip/hip_runtime.h"
#include "MWKernelHeaders.hpp"
#include <math.h>
#include <stdio.h>
 void __global__ __launch_bounds__(1024) scale_scalar_kernel(float* 
inputBuffer, float* outputBuffer, float* olKGEIcsxmLSoMhRhEtP, long int 
YGiQICncmsGZkNUyiQyg) {  for (long int idx = blockDim.x * blockIdx.x + 
threadIdx.x; idx < YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) {  
outputBuffer[idx] = olKGEIcsxmLSoMhRhEtP[0]*inputBuffer[idx]; } } void __global__ 
__launch_bounds__(1024) scale_vector_kernel(float* inputBuffer, float* 
outputBuffer, float* olKGEIcsxmLSoMhRhEtP, double YNDVziqpDddiXQKYZZhX, 
double YMNbgnUYZspjMLjwcIOS, long int YGiQICncmsGZkNUyiQyg) {  for 
(long int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < 
YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) { double batchIdx = 
floor(idx / YMNbgnUYZspjMLjwcIOS); double i_batch = idx - (batchIdx * 
YMNbgnUYZspjMLjwcIOS); double channelIdx = floor(i_batch / 
YNDVziqpDddiXQKYZZhX); outputBuffer[idx] = 
olKGEIcsxmLSoMhRhEtP[static_cast<long int>(channelIdx)]*inputBuffer[idx]; } } void 
__global__ __launch_bounds__(1024) scale_matrix2d_kernel(float* inputBuffer, 
float* outputBuffer, float* olKGEIcsxmLSoMhRhEtP, double 
YNDVziqpDddiXQKYZZhX, long int YGiQICncmsGZkNUyiQyg) {  for (long int 
idx = blockDim.x * blockIdx.x + threadIdx.x; idx < YGiQICncmsGZkNUyiQyg; idx += 
blockDim.x * gridDim.x) { double totalChannelIdx = floor(idx / 
YNDVziqpDddiXQKYZZhX); double i_channel = idx - (totalChannelIdx * 
YNDVziqpDddiXQKYZZhX); outputBuffer[idx] = 
olKGEIcsxmLSoMhRhEtP[static_cast<long int>(i_channel)]*inputBuffer[idx]; } } void 
__global__ __launch_bounds__(1024) scale_tensor3d_kernel(float* inputBuffer, 
float* outputBuffer, float* olKGEIcsxmLSoMhRhEtP, double 
YMNbgnUYZspjMLjwcIOS, long int YGiQICncmsGZkNUyiQyg) {  for (long int 
idx = blockDim.x * blockIdx.x + threadIdx.x; idx < YGiQICncmsGZkNUyiQyg; idx += 
blockDim.x * gridDim.x) { double batchIdx = floor(idx / 
YMNbgnUYZspjMLjwcIOS); double i_batch = idx - (batchIdx * 
YMNbgnUYZspjMLjwcIOS); outputBuffer[idx] = 
olKGEIcsxmLSoMhRhEtP[static_cast<long int>(i_batch)]*inputBuffer[idx]; } }  void 
__global__ __launch_bounds__(1024) offset_scalar_kernel(float* inputBuffer, 
float* outputBuffer, float* fYaOQTeunPwVjnhhTECh, long int YGiQICncmsGZkNUyiQyg, 
bool ZDWLzHUkuZuIUZHfbGDY, int bDTIjtxZiSHtjwzgEluE, int 
unSXtdjDjpysqxmbIiPv) {  for (long int idx = blockDim.x * blockIdx.x + 
threadIdx.x; idx < YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) { float 
out = inputBuffer[idx] + fYaOQTeunPwVjnhhTECh[0]; if (ZDWLzHUkuZuIUZHfbGDY){ out = 
out > unSXtdjDjpysqxmbIiPv ? unSXtdjDjpysqxmbIiPv : out; out = out < 
bDTIjtxZiSHtjwzgEluE ? bDTIjtxZiSHtjwzgEluE : out; } outputBuffer[idx] = out; 
} } void __global__ __launch_bounds__(1024) offset_vector_kernel(float* 
inputBuffer, float* outputBuffer, float* fYaOQTeunPwVjnhhTECh,  double 
YNDVziqpDddiXQKYZZhX, double YMNbgnUYZspjMLjwcIOS, long int 
YGiQICncmsGZkNUyiQyg, bool ZDWLzHUkuZuIUZHfbGDY, int bDTIjtxZiSHtjwzgEluE, int 
unSXtdjDjpysqxmbIiPv) {  for (long int idx = blockDim.x * blockIdx.x + 
threadIdx.x; idx < YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) { 
double batchIdx = floor(idx / YMNbgnUYZspjMLjwcIOS); double i_batch = 
idx - (batchIdx * YMNbgnUYZspjMLjwcIOS); double channelIdx = 
floor(i_batch / YNDVziqpDddiXQKYZZhX); float out = inputBuffer[idx] + 
fYaOQTeunPwVjnhhTECh[static_cast<long int>(channelIdx)]; if 
(ZDWLzHUkuZuIUZHfbGDY){ out = out > unSXtdjDjpysqxmbIiPv ? 
unSXtdjDjpysqxmbIiPv : out; out = out < bDTIjtxZiSHtjwzgEluE ? 
bDTIjtxZiSHtjwzgEluE : out; } outputBuffer[idx] = out; } } void __global__ 
__launch_bounds__(1024) offset_matrix2d_kernel(float* inputBuffer, float* 
outputBuffer, float* fYaOQTeunPwVjnhhTECh, double YNDVziqpDddiXQKYZZhX, 
long int YGiQICncmsGZkNUyiQyg, bool ZDWLzHUkuZuIUZHfbGDY, int 
bDTIjtxZiSHtjwzgEluE, int unSXtdjDjpysqxmbIiPv) {  for (long int idx = 
blockDim.x * blockIdx.x + threadIdx.x; idx < YGiQICncmsGZkNUyiQyg; idx += 
blockDim.x * gridDim.x) { double totalChannelIdx = floor(idx / 
YNDVziqpDddiXQKYZZhX); double i_channel = idx - (totalChannelIdx * 
YNDVziqpDddiXQKYZZhX); float out = inputBuffer[idx] + 
fYaOQTeunPwVjnhhTECh[static_cast<long int>(i_channel)]; if (ZDWLzHUkuZuIUZHfbGDY){ 
out = out > unSXtdjDjpysqxmbIiPv ? unSXtdjDjpysqxmbIiPv : out; out = out < 
bDTIjtxZiSHtjwzgEluE ? bDTIjtxZiSHtjwzgEluE : out; } outputBuffer[idx] = out; 
} } void __global__ __launch_bounds__(1024) offset_tensor3d_kernel(float* 
inputBuffer, float* outputBuffer, float* fYaOQTeunPwVjnhhTECh, double 
YMNbgnUYZspjMLjwcIOS, long int YGiQICncmsGZkNUyiQyg, bool 
ZDWLzHUkuZuIUZHfbGDY, int bDTIjtxZiSHtjwzgEluE, int unSXtdjDjpysqxmbIiPv) {  
for (long int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < 
YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) { double batchIdx = 
floor(idx / YMNbgnUYZspjMLjwcIOS); double i_batch = idx - (batchIdx * 
YMNbgnUYZspjMLjwcIOS); float out = inputBuffer[idx] + 
fYaOQTeunPwVjnhhTECh[static_cast<long int>(i_batch)]; if (ZDWLzHUkuZuIUZHfbGDY){ 
out = out > unSXtdjDjpysqxmbIiPv ? unSXtdjDjpysqxmbIiPv : out; out = out < 
bDTIjtxZiSHtjwzgEluE ? bDTIjtxZiSHtjwzgEluE : out; } outputBuffer[idx] = out; 
} } 