#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: predict.cu
//
// GPU Coder version                    : 2.2
// CUDA/C/C++ source code generated on  : 10-Dec-2021 14:26:27
//

// Include Files
#include "predict.h"
#include "DeepLearningNetwork.h"
#include "myAlexNetGPU_internal_types.h"
#include "MWCudaDimUtility.hpp"

// Type Definitions
struct cell_wrap_12 {
  float f1[1000];
};

struct cell_wrap_9 {
  float f1[154587];
};

// Function Declarations
static __global__ void
DeepLearningNetwork_predict_kernel21(const unsigned char varargin_1[154587],
                                     unsigned char input[154587]);

static __global__ void
DeepLearningNetwork_predict_kernel22(const unsigned char input[154587],
                                     cell_wrap_9 inMiniBatchGroup[1]);

static __global__ void
DeepLearningNetwork_predict_kernel23(const cell_wrap_12 outMiniBatchGroup[1],
                                     float varargout_1[1000]);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char varargin_1[154587]
//                unsigned char input[154587]
// Return Type  : void
//
static __global__
    __launch_bounds__(512, 1) void DeepLearningNetwork_predict_kernel21(
        const unsigned char varargin_1[154587], unsigned char input[154587])
{
  unsigned long threadId;
  int i;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId);
  if (i < 154587) {
    input[i] = varargin_1[i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char input[154587]
//                cell_wrap_9 inMiniBatchGroup[1]
// Return Type  : void
//
static __global__
    __launch_bounds__(512, 1) void DeepLearningNetwork_predict_kernel22(
        const unsigned char input[154587], cell_wrap_9 inMiniBatchGroup[1])
{
  unsigned long threadId;
  int i;
  int i1;
  int p;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId % 227UL);
  threadId = (threadId - static_cast<unsigned long>(i)) / 227UL;
  i1 = static_cast<int>(threadId % 227UL);
  threadId = (threadId - static_cast<unsigned long>(i1)) / 227UL;
  p = static_cast<int>(threadId);
  if ((static_cast<int>((static_cast<int>(p < 3)) &&
                        (static_cast<int>(i1 < 227)))) &&
      (static_cast<int>(i < 227))) {
    inMiniBatchGroup[0].f1[(i + 227 * i1) + 51529 * p] =
        static_cast<float>(input[(i1 + 227 * i) + 51529 * p]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const cell_wrap_12 outMiniBatchGroup[1]
//                float varargout_1[1000]
// Return Type  : void
//
static __global__
    __launch_bounds__(512, 1) void DeepLearningNetwork_predict_kernel23(
        const cell_wrap_12 outMiniBatchGroup[1], float varargout_1[1000])
{
  unsigned long threadId;
  int i;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId);
  if (i < 1000) {
    varargout_1[i] = outMiniBatchGroup[0].f1[i];
  }
}

//
// Arguments    : alexnet0_0 *obj
//                const unsigned char varargin_1[154587]
//                float varargout_1[1000]
// Return Type  : void
//
namespace coder {
void DeepLearningNetwork_predict(alexnet0_0 *obj,
                                 const unsigned char varargin_1[154587],
                                 float varargout_1[1000])
{
  cell_wrap_12(*gpu_outMiniBatchGroup)[1];
  cell_wrap_9(*gpu_inMiniBatchGroup)[1];
  float(*gpu_varargout_1)[1000];
  unsigned char(*gpu_input)[154587];
  unsigned char(*gpu_varargin_1)[154587];
  hipMalloc(&gpu_varargout_1, 4000UL);
  hipMalloc(&gpu_outMiniBatchGroup, 4000UL);
  hipMalloc(&gpu_inMiniBatchGroup, 618348UL);
  hipMalloc(&gpu_input, 154587UL);
  hipMalloc(&gpu_varargin_1, 154587UL);
  hipMemcpy(*gpu_varargin_1, varargin_1, 154587UL, hipMemcpyHostToDevice);
  DeepLearningNetwork_predict_kernel21<<<dim3(302U, 1U, 1U),
                                         dim3(512U, 1U, 1U)>>>(*gpu_varargin_1,
                                                               *gpu_input);
  DeepLearningNetwork_predict_kernel22<<<dim3(302U, 1U, 1U),
                                         dim3(512U, 1U, 1U)>>>(
      *gpu_input, *gpu_inMiniBatchGroup);
  hipMemcpy(obj->getInputDataPointer(0), (*gpu_inMiniBatchGroup)[0].f1,
             obj->getLayerOutputSize(0, 0), hipMemcpyDeviceToDevice);
  obj->predict();
  hipMemcpy((*gpu_outMiniBatchGroup)[0].f1, obj->getLayerOutput(18, 0),
             obj->getLayerOutputSize(18, 0), hipMemcpyDeviceToDevice);
  DeepLearningNetwork_predict_kernel23<<<dim3(2U, 1U, 1U),
                                         dim3(512U, 1U, 1U)>>>(
      *gpu_outMiniBatchGroup, *gpu_varargout_1);
  hipMemcpy(varargout_1, *gpu_varargout_1, 4000UL, hipMemcpyDeviceToHost);
  hipFree(*gpu_varargin_1);
  hipFree(*gpu_input);
  hipFree(*gpu_inMiniBatchGroup);
  hipFree(*gpu_outMiniBatchGroup);
  hipFree(*gpu_varargout_1);
}

} // namespace coder

//
// File trailer for predict.cu
//
// [EOF]
//
