#include "MWCudnnReLULayerImpl.hpp"
#include "MWCudnnCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
 namespace MWCudnnTarget { MWReLULayerImpl::MWReLULayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl) { 
CUDNN_CALL(hipdnnCreateActivationDescriptor(&muwRQxtWMMXAPxSuMYBw)); 
createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex()); } 
MWReLULayerImpl::~MWReLULayerImpl() { } void MWReLULayerImpl::propagateSize() { 
MWTensorBase* opTensor = getLayer()->getOutputTensor(0); 
hipdnnTensorDescriptor_t* desc = getDescriptor(opTensor->getSourcePortIndex()); 
assert(desc); setDescriptor<float>(*desc, 
static_cast<MWTensor<float>*>(opTensor)); 
CUDNN_CALL(hipdnnSetActivationDescriptor(muwRQxtWMMXAPxSuMYBw, 
HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0)); } void 
MWReLULayerImpl::predict() { MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor(0); MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
hipdnnTensorDescriptor_t ipDesc = 
MWCNNLayerImpl::getCuDNNDescriptor(ipTensorBase); 
CUDNN_CALL(hipdnnActivationForward(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
muwRQxtWMMXAPxSuMYBw, getOnePtr(), ipDesc, ipTensor->getData(), getZeroPtr(), 
*desc, opTensor->getData())); } void MWReLULayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyActivationDescriptor(muwRQxtWMMXAPxSuMYBw)); } } 