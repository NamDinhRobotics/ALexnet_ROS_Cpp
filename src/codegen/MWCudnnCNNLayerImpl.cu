#include "hip/hip_runtime.h"
#include "MWCudnnCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include <cstdlib>
#include <cassert>
#include <cstdio>
#include <stdexcept>
#include <sstream>
 namespace MWCudnnTarget { float* malloc_call_line_file(size_t msize, const int 
ZqQxEyCjEixByRZYMkbj, const char *PfisSEEWDaQFynnzlcin) { float * mem = 
(float*)malloc(msize); if (!mem) { 
MWCNNLayerImpl::throwAllocationError(ZqQxEyCjEixByRZYMkbj, PfisSEEWDaQFynnzlcin); }  
return mem; } void cuda_call_line_file(hipError_t rMMjgjGRAiLVlTlRSByU, const int 
ZqQxEyCjEixByRZYMkbj, const char *PfisSEEWDaQFynnzlcin) { if (rMMjgjGRAiLVlTlRSByU != 
hipSuccess) { throw_cuda_error(rMMjgjGRAiLVlTlRSByU, ZqQxEyCjEixByRZYMkbj, 
PfisSEEWDaQFynnzlcin);  } } void throw_cuda_error(hipError_t rMMjgjGRAiLVlTlRSByU, 
const int ZqQxEyCjEixByRZYMkbj, const char *PfisSEEWDaQFynnzlcin) { std::stringstream ss; 
ss << "CUDA Error " << rMMjgjGRAiLVlTlRSByU << "(" << 
hipGetErrorString(rMMjgjGRAiLVlTlRSByU) << ") at line " << ZqQxEyCjEixByRZYMkbj << 
", file: " << PfisSEEWDaQFynnzlcin << "\n";  rMMjgjGRAiLVlTlRSByU = hipGetLastError(); 
 throw std::runtime_error(ss.str());  } void cudnn_call_line_file(hipdnnStatus_t 
rMMjgjGRAiLVlTlRSByU, const int ZqQxEyCjEixByRZYMkbj, const char *PfisSEEWDaQFynnzlcin) { if 
(rMMjgjGRAiLVlTlRSByU != HIPDNN_STATUS_SUCCESS) {  std::stringstream ss; ss << 
"CuDNN Error " << rMMjgjGRAiLVlTlRSByU << "(" << 
hipdnnGetErrorString(rMMjgjGRAiLVlTlRSByU) << ") at line " << ZqQxEyCjEixByRZYMkbj << 
", file: " << PfisSEEWDaQFynnzlcin << "\n"; throw std::runtime_error(ss.str()); } } 
const char* cublasGetErrorString(hipblasStatus_t rMMjgjGRAiLVlTlRSByU) { 
switch(rMMjgjGRAiLVlTlRSByU) { case HIPBLAS_STATUS_SUCCESS: return 
"HIPBLAS_STATUS_SUCCESS"; case HIPBLAS_STATUS_NOT_INITIALIZED: return 
"HIPBLAS_STATUS_NOT_INITIALIZED"; case HIPBLAS_STATUS_ALLOC_FAILED: return 
"HIPBLAS_STATUS_ALLOC_FAILED"; case HIPBLAS_STATUS_INVALID_VALUE: return 
"HIPBLAS_STATUS_INVALID_VALUE";  case HIPBLAS_STATUS_ARCH_MISMATCH: return 
"HIPBLAS_STATUS_ARCH_MISMATCH";  case HIPBLAS_STATUS_MAPPING_ERROR: return 
"HIPBLAS_STATUS_MAPPING_ERROR"; case HIPBLAS_STATUS_EXECUTION_FAILED: return 
"HIPBLAS_STATUS_EXECUTION_FAILED";  case HIPBLAS_STATUS_INTERNAL_ERROR: return 
"HIPBLAS_STATUS_INTERNAL_ERROR";  case HIPBLAS_STATUS_NOT_SUPPORTED: return 
"HIPBLAS_STATUS_NOT_SUPPORTED";  case HIPBLAS_STATUS_UNKNOWN: return 
"HIPBLAS_STATUS_UNKNOWN";  } return "unknown error"; } void 
cublas_call_line_file(hipblasStatus_t rMMjgjGRAiLVlTlRSByU, const int 
ZqQxEyCjEixByRZYMkbj, const char *PfisSEEWDaQFynnzlcin) { if (rMMjgjGRAiLVlTlRSByU != 
HIPBLAS_STATUS_SUCCESS) { std::stringstream ss; ss << "CuBLAS Error " << 
rMMjgjGRAiLVlTlRSByU << "(" << cublasGetErrorString(rMMjgjGRAiLVlTlRSByU) << 
") at line " << ZqQxEyCjEixByRZYMkbj << ", file: " << PfisSEEWDaQFynnzlcin << "\n";  
throw std::runtime_error(ss.str()); } } 
MWCNNLayerImpl::MWCNNLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl) : MWCNNLayerImplBase(layer) , OwscQfaoXJuSJFwXQahz(0.0) , 
OumvfgWXDdmsQaciHMHx(1.0) , OVOphSOolqRQDDoKPwxy(-1.0) , 
cQBKlCKXxecGPJrXBXdk(ntwk_impl) { } MWCNNLayerImpl::~MWCNNLayerImpl() { 
for(std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
jhFUWlztBndwjbXwYNaJ.begin(); it != jhFUWlztBndwjbXwYNaJ.end(); ++it) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*it->second)); delete it->second; 
it->second = 0; } } void MWCNNLayerImpl::allocateOutput(int outIdx) { if 
(getLayer()->getOutputTensor(outIdx)->isFloat()) { 
this->allocateOutputImpl<float>(outIdx); } else { 
assert(getLayer()->getOutputTensor(outIdx)->isInt8()); 
this->allocateOutputImpl<signed char>(outIdx); } } template <class T> void 
MWCNNLayerImpl::allocateOutputImpl(int outIdx) { MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(outIdx); bool bufferReuse = 
opTensorBase->getopBufIndex() >= 0; if (bufferReuse) { 
assert(opTensorBase->isFloat()); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); float* buffer = 
cQBKlCKXxecGPJrXBXdk->getBufferPtr(opTensor->getopBufIndex()); int inIdx = 
getLayer()->getInPlaceIndex(outIdx); if (inIdx != -1) { MWTensor<float>* 
ipTensor = static_cast<MWTensor<float>*>(getLayer()->getInputTensor(inIdx)); 
buffer = ipTensor->getData(); assert(buffer); } float* opPointer = 
offsetOutputPointer(buffer, outIdx); assert(opPointer); 
opTensor->setData(opPointer); } else { int inIdx = 
getLayer()->getInPlaceIndex(outIdx); if (inIdx != -1) { MWTensor<T>* ipTensor = 
static_cast<MWTensor<T>*>(getLayer()->getInputTensor(inIdx)); MWTensor<T>* 
opTensor = static_cast<MWTensor<T>*>(opTensorBase); T* ipData = 
ipTensor->getData(); assert(ipData); T* opPointer = offsetOutputPointer(ipData, 
outIdx); assert(opPointer); opTensor->setData(opPointer); } else { MWTensor<T>* 
opTensor = static_cast<MWTensor<T>*>(opTensorBase); T* NzudlCvUcxBgCSkidIap;  
CUDA_CALL(hipMalloc((void**)&NzudlCvUcxBgCSkidIap, 
sizeof(T)*opTensor->getNumElements())); opTensor->setData(NzudlCvUcxBgCSkidIap); } } 
} template void MWCNNLayerImpl::allocateOutputImpl<float>(int); template void 
MWCNNLayerImpl::allocateOutputImpl<signed char>(int); void 
MWCNNLayerImpl::deallocateOutput(int outIdx) { if 
(getLayer()->getOutputTensor(outIdx)->isFloat()) { 
this->deallocateOutputImpl<float>(outIdx); } else { 
assert(getLayer()->getOutputTensor(outIdx)->isInt8()); 
this->deallocateOutputImpl<signed char>(outIdx); } } template <class T> void 
MWCNNLayerImpl::deallocateOutputImpl(int outIdx) { 
if(getLayer()->getInPlaceIndex(outIdx) == -1) { MWTensor<T>* opTensor = 
static_cast<MWTensor<T>*>(getLayer()->getOutputTensor(outIdx)); T* data = 
opTensor->getData(); CUDA_FREE_CALL(data); } } template void 
MWCNNLayerImpl::deallocateOutputImpl<float>(int); template void 
MWCNNLayerImpl::deallocateOutputImpl<signed char>(int); float* 
MWCNNLayerImpl::getZeroPtr() { return &OwscQfaoXJuSJFwXQahz; } float* 
MWCNNLayerImpl::getOnePtr() { return &OumvfgWXDdmsQaciHMHx; } float* 
MWCNNLayerImpl::getNegOnePtr() { return &OVOphSOolqRQDDoKPwxy; } 
hipdnnTensorDescriptor_t* MWCNNLayerImpl::createAndAddDescriptor(int index) { 
std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
jhFUWlztBndwjbXwYNaJ.find(index); assert(it == jhFUWlztBndwjbXwYNaJ.end()); 
hipdnnTensorDescriptor_t* newDescriptor = new hipdnnTensorDescriptor_t; if 
(!newDescriptor) { MWCNNLayerImpl::throwAllocationError(__LINE__ , __FILE__); } 
jhFUWlztBndwjbXwYNaJ[index] = newDescriptor; 
CUDNN_CALL(hipdnnCreateTensorDescriptor(newDescriptor)); return newDescriptor; } 
hipdnnTensorDescriptor_t* MWCNNLayerImpl::getDescriptor(int index) {  
std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
jhFUWlztBndwjbXwYNaJ.find(index); if (it != jhFUWlztBndwjbXwYNaJ.end()) { 
return it->second; } else { return NULL; } } template <class T> void 
MWCNNLayerImpl::setDescriptor(hipdnnTensorDescriptor_t& desc, MWTensor<T>* 
tensor) { if (tensor->getSequenceLength() == 1) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, 
MWCNNLayerImpl::getCuDNNDataType<T>(), tensor->getBatchSize(),  
tensor->getChannels(),  tensor->getHeight(),  tensor->getWidth()));  } else { 
int dims[5] = {tensor->getSequenceLength(), tensor->getBatchSize(), 
tensor->getChannels(), tensor->getHeight(), tensor->getWidth()}; int 
strides[5]; MWTensorBase::getStrides(dims, 5, strides); 
CUDNN_CALL(hipdnnSetTensorNdDescriptor(desc, 
MWCNNLayerImpl::getCuDNNDataType<T>(), 5, dims, strides)); }  } template void 
MWCNNLayerImpl::setDescriptor<float>(hipdnnTensorDescriptor_t&, 
MWTensor<float>*); template void MWCNNLayerImpl::setDescriptor<signed 
char>(hipdnnTensorDescriptor_t&, MWTensor<signed char>*); template <> 
hipdnnDataType_t MWCNNLayerImpl::getCuDNNDataType<float>() { return 
HIPDNN_DATA_FLOAT; } template <> hipdnnDataType_t 
MWCNNLayerImpl::getCuDNNDataType<signed char>() { return HIPDNN_DATA_INT8; } 
hipdnnTensorDescriptor_t MWCNNLayerImpl::getCuDNNDescriptor(MWTensorBase* 
tensor) { MWCNNLayer* layer = tensor->getOwner(); MWCNNLayerImpl* impl = 
static_cast<MWCNNLayerImpl*>(layer->getImpl()); if(impl) { 
hipdnnTensorDescriptor_t* desc = 
impl->getDescriptor(tensor->getSourcePortIndex()); if (desc == NULL) { 
impl->createAndAddDescriptor(tensor->getSourcePortIndex()); desc = 
impl->getDescriptor(tensor->getSourcePortIndex()); assert(desc);  } if 
(tensor->isFloat()) { MWCNNLayerImpl::setDescriptor<float>(*desc, 
static_cast<MWTensor<float>*>(tensor)); } else { assert(tensor->isInt8()); 
MWCNNLayerImpl::setDescriptor<signed char>(*desc, static_cast<MWTensor<signed 
char>*>(tensor)); } return *desc; } else { hipdnnTensorDescriptor_t 
tmpDescriptor; CUDNN_CALL(hipdnnCreateTensorDescriptor(&tmpDescriptor)); if 
(tensor->isFloat()) { MWCNNLayerImpl::setDescriptor<float>(tmpDescriptor, 
static_cast<MWTensor<float>*>(tensor)); } else { assert(tensor->isInt8()); 
MWCNNLayerImpl::setDescriptor<signed char>(tmpDescriptor, 
static_cast<MWTensor<signed char>*>(tensor)); } return tmpDescriptor; } } void 
__global__ __launch_bounds__(1024) padInputImpl(float* in, int inputH, int 
inputW, int inputCh, int outputH, int outputW, int offsetH, int offsetW, float* 
out, int inputElems) { for(int i = blockDim.x * blockIdx.x + threadIdx.x; i < 
inputElems; i+= blockDim.x*gridDim.x) { int idxB = i/(inputH*inputW*inputCh); 
int rem = (i - idxB*(inputH*inputW*inputCh)); int idxCh = rem/(inputH*inputW); 
int rem1 = rem - idxCh*(inputH*inputW); int idxH = rem1/inputW; int idxCol = 
rem1 - idxH*inputW; if ((idxH < inputH) && (idxCol < inputW)) { int outputR = 
idxH + offsetH; int outputCol = idxCol + offsetW; int outputCh = inputCh; 
out[idxB*(outputH*outputW*outputCh) + idxCh*(outputH*outputW) + 
outputR*(outputW) + outputCol] = in[i]; } } } void 
MWCNNLayerImpl::padInput(float* SugesRlPIbOVzRgNWRnl, int UzaGmBLFEwmwaFXebUma, int 
VFKMunbyHoAmpHUSkuUn, int UzOdnHgHuNHtprVxxxXl, int jmcFOAbZArjGDNhshSro, int 
kFQQPKSOkZeHlmrkAXuE, int fjfzkUfcCOqjrkAVGfuc, int gGtqPpFypRDdniDkzLXN, float* 
iwclITrbVyVrJaArrXNr, int dkLDkRwCBjeybwDHbKiE) { int shEncNmxJsMuJKwbrwok = 
(dkLDkRwCBjeybwDHbKiE + 31)/32 * 32; shEncNmxJsMuJKwbrwok = 
(shEncNmxJsMuJKwbrwok < 1024) ? shEncNmxJsMuJKwbrwok : 1024; int 
KHClOltUSuqFVVErSxVb = (dkLDkRwCBjeybwDHbKiE + shEncNmxJsMuJKwbrwok - 
1)/shEncNmxJsMuJKwbrwok; padInputImpl<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>(SugesRlPIbOVzRgNWRnl, UzaGmBLFEwmwaFXebUma, 
VFKMunbyHoAmpHUSkuUn, UzOdnHgHuNHtprVxxxXl, jmcFOAbZArjGDNhshSro, kFQQPKSOkZeHlmrkAXuE, 
fjfzkUfcCOqjrkAVGfuc, gGtqPpFypRDdniDkzLXN, iwclITrbVyVrJaArrXNr, dkLDkRwCBjeybwDHbKiE); } 
void __global__ __launch_bounds__(1024) fillOutputBufferImpl(signed char* in, 
int inputH, int inputW, int inputCh, int outputH, int outputW, int offsetH, int 
offsetW, signed char* out, int inputElems, int outputCh) { for(int i = 
blockDim.x * blockIdx.x + threadIdx.x; i < inputElems; i+= 
blockDim.x*gridDim.x) { int idxB = i/(inputH*inputW*inputCh); int rem = (i - 
idxB*(inputH*inputW*inputCh)); int idxCh = rem/(inputH*inputW); int rem1 = rem 
- idxCh*(inputH*inputW); int idxH = rem1/inputW; int idxCol = rem1 - 
idxH*inputW; if ((idxH < inputH) && (idxCol < inputW)) { int outputR = idxH + 
offsetH; int outputCol = idxCol + offsetW; *(out + 
idxB*(outputH*outputW*outputCh) + idxCh*(outputH*outputW) + outputR*(outputW) + 
outputCol) = *(in + i); } } } void MWCNNLayerImpl::fillOutputBuffer(signed 
char* SugesRlPIbOVzRgNWRnl, int UzaGmBLFEwmwaFXebUma, int VFKMunbyHoAmpHUSkuUn, int 
UzOdnHgHuNHtprVxxxXl, int jmcFOAbZArjGDNhshSro, int kFQQPKSOkZeHlmrkAXuE, int 
fjfzkUfcCOqjrkAVGfuc, int gGtqPpFypRDdniDkzLXN, signed char* iwclITrbVyVrJaArrXNr, int 
dkLDkRwCBjeybwDHbKiE, int jfkhqXBmwICFStMidrQt) { int shEncNmxJsMuJKwbrwok 
= (dkLDkRwCBjeybwDHbKiE < 1024) ? dkLDkRwCBjeybwDHbKiE : 1024; int 
KHClOltUSuqFVVErSxVb = (dkLDkRwCBjeybwDHbKiE + shEncNmxJsMuJKwbrwok - 
1)/shEncNmxJsMuJKwbrwok; fillOutputBufferImpl<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>(SugesRlPIbOVzRgNWRnl, UzaGmBLFEwmwaFXebUma, 
VFKMunbyHoAmpHUSkuUn, UzOdnHgHuNHtprVxxxXl, jmcFOAbZArjGDNhshSro, kFQQPKSOkZeHlmrkAXuE, 
fjfzkUfcCOqjrkAVGfuc, gGtqPpFypRDdniDkzLXN, iwclITrbVyVrJaArrXNr, dkLDkRwCBjeybwDHbKiE, 
jfkhqXBmwICFStMidrQt); } void MWCNNLayerImpl::throwAllocationError(const int 
line, const char * file) { std::stringstream ss; ss << 
"Failed to allocate memory at line : " << line << ", file : " << file << "\n"; 
throw std::runtime_error(ss.str());  } void 
MWCNNLayerImpl::setDescriptorForINT8(hipdnnTensorDescriptor_t& desc, 
MWTensor<signed char>* tensor, hipdnnDataType_t dataType, hipdnnTensorFormat_t 
dataFormat) { CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, dataFormat, dataType, 
tensor->getBatchSize(),  tensor->getChannels(),  tensor->getHeight(),  
tensor->getWidth()));  } } 