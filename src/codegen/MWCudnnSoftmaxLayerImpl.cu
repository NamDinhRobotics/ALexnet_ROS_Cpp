#include "MWCudnnSoftmaxLayerImpl.hpp"
#include "MWCudnnCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
 namespace MWCudnnTarget { MWSoftmaxLayerImpl::MWSoftmaxLayerImpl(MWCNNLayer* 
layer, MWTargetNetworkImpl* ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl)  { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&rytJDHzuydvYOLNNROYf)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&sFIUeCwGDlfadqOrGZHC)); } 
MWSoftmaxLayerImpl::~MWSoftmaxLayerImpl() { } void 
MWSoftmaxLayerImpl::propagateSize() { MWCNNLayer* sfmxLayer = getLayer(); 
MWTensorBase* ipTensor = sfmxLayer->getInputTensor(0); MWTensorBase* opTensor = 
sfmxLayer->getOutputTensor(0); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(rytJDHzuydvYOLNNROYf, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, ipTensor->getSequenceLength()*ipTensor->getBatchSize(), 
ipTensor->getChannels(), ipTensor->getHeight(), ipTensor->getWidth())); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(sFIUeCwGDlfadqOrGZHC, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, opTensor->getSequenceLength()*opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); } void 
MWSoftmaxLayerImpl::predict() { MWCNNLayer* sfmxLayer = getLayer(); 
MWTensorBase* ipTensorBase = sfmxLayer->getInputTensor(0); MWTensorBase* 
opTensorBase = sfmxLayer->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); 
CUDNN_CALL(hipdnnSoftmaxForward(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, getOnePtr(), 
rytJDHzuydvYOLNNROYf, ipTensor->getData(), getZeroPtr(), 
sFIUeCwGDlfadqOrGZHC, opTensor->getData())); } void 
MWSoftmaxLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(rytJDHzuydvYOLNNROYf)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(sFIUeCwGDlfadqOrGZHC)); } } 