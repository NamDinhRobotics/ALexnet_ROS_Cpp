#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: myAlexNetGPU.cu
//
// GPU Coder version                    : 2.2
// CUDA/C/C++ source code generated on  : 10-Dec-2021 14:26:27
//

// Include Files
#include "myAlexNetGPU.h"
#include "DeepLearningNetwork.h"
#include "myAlexNetGPU_data.h"
#include "myAlexNetGPU_initialize.h"
#include "myAlexNetGPU_internal_types.h"
#include "predict.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include <cmath>

// Variable Definitions
static alexnet0_0 net;

static bool net_not_empty;

// Function Declarations
static __global__ void
myAlexNetGPU_kernel1(const unsigned char I_data[], const int I_size[3],
                     const int img_size[3], const int b_I_size,
                     const int in_rows, unsigned char img_data[2211840]);

static __global__ void myAlexNetGPU_kernel10(const int plast,
                                             double ipColIndices_data[4313]);

static __global__ void myAlexNetGPU_kernel11(const int plast,
                                             double colWeights_data[4313]);

static __global__ void myAlexNetGPU_kernel12(const double aux2_data[2048],
                                             const int in_cols,
                                             const double scale_idx_1,
                                             const double kwidthCol,
                                             double ipColIndices_data[4313],
                                             double colWeights_data[4313]);

static __global__ void myAlexNetGPU_kernel13(const double rowWeights_data[2951],
                                             double rowWeightsTotal[227]);

static __global__ void myAlexNetGPU_kernel14(const double rowWeights_data[2951],
                                             const int in_rows,
                                             double rowWeightsTotal[227]);

static __global__ void myAlexNetGPU_kernel15(const double colWeights_data[4313],
                                             double colWeightsTotal[227]);

static __global__ void myAlexNetGPU_kernel16(const double colWeights_data[4313],
                                             const int in_rows,
                                             double colWeightsTotal[227]);

static __global__ void myAlexNetGPU_kernel17(
    const double colWeightsTotal[227], const double colWeights_data[4313],
    const unsigned char img_data[2211840], const double ipColIndices_data[4313],
    const int img_size[3], const int partialResize_size[3],
    const double kwidthCol, const int plast,
    unsigned char partialResize_data[490320]);

static __global__ void myAlexNetGPU_kernel18(
    const double rowWeightsTotal[227], const double rowWeights_data[2951],
    const unsigned char partialResize_data[490320],
    const int partialResize_size[3], const double ipRowIndices_data[2951],
    const double kwidthRow, unsigned char out[154587]);

static __global__ void myAlexNetGPU_kernel19(
    const double rowWeightsTotal[227], const double rowWeights_data[2951],
    const unsigned char img_data[2211840], const int img_size[3],
    const double ipRowIndices_data[2951], const int partialResize_size[3],
    const double kwidthRow, const int plast,
    unsigned char partialResize_data[697344]);

static __global__ void myAlexNetGPU_kernel2(const unsigned char I_data[],
                                            const int I_size,
                                            unsigned char img_data[2211840]);

static __global__ void myAlexNetGPU_kernel20(
    const double colWeightsTotal[227], const double colWeights_data[4313],
    const unsigned char partialResize_data[697344],
    const int partialResize_size[3], const double ipColIndices_data[4313],
    const double kwidthCol, unsigned char out[154587]);

static __global__ void
myAlexNetGPU_kernel3(const unsigned char img_data[2211840],
                     const int img_size[3], const int b_img_size[3],
                     const int plast, const int in_rows,
                     unsigned char b_img_data[2211840]);

static __global__ void
myAlexNetGPU_kernel4(const unsigned char img_data[2211840], const int img_size,
                     unsigned char b_img_data[2211840]);

static __global__ void myAlexNetGPU_kernel5(const int in_rows, const int plast,
                                            double aux1_data[1440]);

static __global__ void myAlexNetGPU_kernel6(const int in_cols, const int plast,
                                            double aux2_data[2048]);

static __global__ void myAlexNetGPU_kernel7(const int plast,
                                            double ipRowIndices_data[2951]);

static __global__ void myAlexNetGPU_kernel8(const int plast,
                                            double rowWeights_data[2951]);

static __global__ void myAlexNetGPU_kernel9(const double aux1_data[1440],
                                            const int in_rows,
                                            const double scale_idx_0,
                                            const double kwidthRow,
                                            double ipRowIndices_data[2951],
                                            double rowWeights_data[2951]);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char I_data[]
//                const int I_size[3]
//                const int img_size[3]
//                const int b_I_size
//                const int in_rows
//                unsigned char img_data[2211840]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void myAlexNetGPU_kernel1(
    const unsigned char I_data[], const int I_size[3], const int img_size[3],
    const int b_I_size, const int in_rows, unsigned char img_data[2211840])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = 3UL * ((static_cast<unsigned long>(b_I_size) + 1UL) *
                   (static_cast<unsigned long>(in_rows - 1) + 1UL)) -
            1UL;
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    unsigned long tmpIndex;
    int colIndices;
    int k;
    int rowIndices;
    colIndices = static_cast<int>(idx % 3UL);
    tmpIndex = (idx - static_cast<unsigned long>(colIndices)) / 3UL;
    rowIndices = static_cast<int>(tmpIndex %
                                  (static_cast<unsigned long>(b_I_size) + 1UL));
    tmpIndex = (tmpIndex - static_cast<unsigned long>(rowIndices)) /
               (static_cast<unsigned long>(b_I_size) + 1UL);
    k = static_cast<int>(tmpIndex);
    img_data[((static_cast<int>(static_cast<short>(k + 1)) +
               img_size[0] *
                   (static_cast<int>(static_cast<short>(rowIndices + 1)) - 1)) +
              img_size[0] * img_size[1] * colIndices) -
             1] = I_data
        [(colIndices +
          3 * (static_cast<int>(static_cast<short>(rowIndices + 1)) - 1)) +
         3 * I_size[1] * (static_cast<int>(static_cast<short>(k + 1)) - 1)];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int plast
//                double ipColIndices_data[4313]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void myAlexNetGPU_kernel10(
    const int plast, double ipColIndices_data[4313])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(227 * plast - 1);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    ipColIndices_data[oldIdx] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int plast
//                double colWeights_data[4313]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void myAlexNetGPU_kernel11(
    const int plast, double colWeights_data[4313])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(227 * plast - 1);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    colWeights_data[oldIdx] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double aux2_data[2048]
//                const int in_cols
//                const double scale_idx_1
//                const double kwidthCol
//                double ipColIndices_data[4313]
//                double colWeights_data[4313]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void myAlexNetGPU_kernel12(
    const double aux2_data[2048], const int in_cols, const double scale_idx_1,
    const double kwidthCol, double ipColIndices_data[4313],
    double colWeights_data[4313])
{
  unsigned long threadId;
  int colIdx;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  colIdx = static_cast<int>(threadId);
  if (colIdx < 227) {
    int rowIndices;
    rowIndices = static_cast<int>(ceil(kwidthCol));
    for (int k{0}; k < rowIndices; k++) {
      double absx;
      double absx2;
      double sumVal;
      int b_y;
      int colIndices;
      int l;
      sumVal = (static_cast<double>(colIdx) + 1.0) / scale_idx_1 +
               0.5 * (1.0 - 1.0 / scale_idx_1);
      colIndices = static_cast<int>(floor(sumVal - kwidthCol / 2.0));
      sumVal -= static_cast<double>(colIndices + k) + 1.0;
      if (scale_idx_1 < 1.0) {
        sumVal *= scale_idx_1;
      }
      absx = fabs(sumVal);
      absx2 = absx * absx;
      sumVal = pow(absx, 3.0);
      sumVal = ((1.5 * sumVal - 2.5 * absx2) + 1.0) *
                   static_cast<double>(absx <= 1.0) +
               (((-0.5 * sumVal + 2.5 * absx2) - 4.0 * absx) + 2.0) *
                   static_cast<double>((static_cast<int>(1.0 < absx)) &&
                                       (static_cast<int>(absx <= 2.0)));
      if (scale_idx_1 < 1.0) {
        colWeights_data[colIdx + 227 * k] = scale_idx_1 * sumVal;
      } else {
        colWeights_data[colIdx + 227 * k] = sumVal;
      }
      b_y = in_cols << 1;
      colIndices = (colIndices + k) + 1;
      l = colIndices - 1;
      if (b_y == 0) {
        if (colIndices - 1 == 0) {
          l = 0;
        }
      } else if (colIndices - 1 == 0) {
        l = 0;
      } else {
        l = static_cast<int>(fmod(static_cast<double>(colIndices) - 1.0,
                                  static_cast<double>(b_y)));
        if ((static_cast<int>(l != 0)) &&
            (static_cast<int>(colIndices - 1 < 0))) {
          l += b_y;
        }
      }
      ipColIndices_data[colIdx + 227 * k] = aux2_data[l];
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double rowWeights_data[2951]
//                double rowWeightsTotal[227]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void myAlexNetGPU_kernel13(
    const double rowWeights_data[2951], double rowWeightsTotal[227])
{
  unsigned long threadId;
  int colIndices;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  colIndices = static_cast<int>(threadId);
  if (colIndices < 227) {
    rowWeightsTotal[colIndices] = rowWeights_data[colIndices];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double rowWeights_data[2951]
//                const int in_rows
//                double rowWeightsTotal[227]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void myAlexNetGPU_kernel14(
    const double rowWeights_data[2951], const int in_rows,
    double rowWeightsTotal[227])
{
  unsigned long threadId;
  int colIndices;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  colIndices = static_cast<int>(threadId);
  if (colIndices < 227) {
    rowWeightsTotal[colIndices] += rowWeights_data[in_rows + colIndices];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double colWeights_data[4313]
//                double colWeightsTotal[227]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void myAlexNetGPU_kernel15(
    const double colWeights_data[4313], double colWeightsTotal[227])
{
  unsigned long threadId;
  int colIndices;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  colIndices = static_cast<int>(threadId);
  if (colIndices < 227) {
    colWeightsTotal[colIndices] = colWeights_data[colIndices];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double colWeights_data[4313]
//                const int in_rows
//                double colWeightsTotal[227]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void myAlexNetGPU_kernel16(
    const double colWeights_data[4313], const int in_rows,
    double colWeightsTotal[227])
{
  unsigned long threadId;
  int colIndices;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  colIndices = static_cast<int>(threadId);
  if (colIndices < 227) {
    colWeightsTotal[colIndices] += colWeights_data[in_rows + colIndices];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double colWeightsTotal[227]
//                const double colWeights_data[4313]
//                const unsigned char img_data[2211840]
//                const double ipColIndices_data[4313]
//                const int img_size[3]
//                const int partialResize_size[3]
//                const double kwidthCol
//                const int plast
//                unsigned char partialResize_data[490320]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void myAlexNetGPU_kernel17(
    const double colWeightsTotal[227], const double colWeights_data[4313],
    const unsigned char img_data[2211840], const double ipColIndices_data[4313],
    const int img_size[3], const int partialResize_size[3],
    const double kwidthCol, const int plast,
    unsigned char partialResize_data[490320])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = 3UL * (227UL * (static_cast<unsigned long>(plast - 1) + 1UL)) - 1UL;
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    double sumVal;
    unsigned long tmpIndex;
    int colIdx;
    int colIndices;
    int rowIdx;
    int rowIndices;
    unsigned char u;
    rowIndices = static_cast<int>(idx % 3UL);
    tmpIndex = (idx - static_cast<unsigned long>(rowIndices)) / 3UL;
    colIdx = static_cast<int>(tmpIndex % 227UL);
    tmpIndex = (tmpIndex - static_cast<unsigned long>(colIdx)) / 227UL;
    rowIdx = static_cast<int>(tmpIndex);
    sumVal = 0.0;
    colIndices = static_cast<int>(ceil(kwidthCol));
    for (int l{0}; l < colIndices; l++) {
      sumVal +=
          static_cast<double>(
              img_data[(rowIdx + img_size[0] *
                                     (static_cast<int>(
                                          ipColIndices_data[colIdx + 227 * l]) -
                                      1)) +
                       img_size[0] * img_size[1] * rowIndices]) *
          (colWeights_data[colIdx + 227 * l] / colWeightsTotal[colIdx]);
    }
    sumVal = round(sumVal);
    if (sumVal < 256.0) {
      if (sumVal >= 0.0) {
        u = static_cast<unsigned char>(sumVal);
      } else {
        u = static_cast<unsigned char>(0U);
      }
    } else if (sumVal >= 256.0) {
      u = MAX_uint8_T;
    } else {
      u = static_cast<unsigned char>(0U);
    }
    partialResize_data[(rowIdx + partialResize_size[0] * colIdx) +
                       partialResize_size[0] * 227 * rowIndices] = u;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double rowWeightsTotal[227]
//                const double rowWeights_data[2951]
//                const unsigned char partialResize_data[490320]
//                const int partialResize_size[3]
//                const double ipRowIndices_data[2951]
//                const double kwidthRow
//                unsigned char out[154587]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void myAlexNetGPU_kernel18(
    const double rowWeightsTotal[227], const double rowWeights_data[2951],
    const unsigned char partialResize_data[490320],
    const int partialResize_size[3], const double ipRowIndices_data[2951],
    const double kwidthRow, unsigned char out[154587])
{
  unsigned long threadId;
  int colIdx;
  int rowIdx;
  int rowIndices;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  rowIndices = static_cast<int>(threadId % 3UL);
  threadId = (threadId - static_cast<unsigned long>(rowIndices)) / 3UL;
  rowIdx = static_cast<int>(threadId % 227UL);
  threadId = (threadId - static_cast<unsigned long>(rowIdx)) / 227UL;
  colIdx = static_cast<int>(threadId);
  if ((static_cast<int>((static_cast<int>(colIdx < 227)) &&
                        (static_cast<int>(rowIdx < 227)))) &&
      (static_cast<int>(rowIndices < 3))) {
    double sumVal;
    int colIndices;
    unsigned char u;
    sumVal = 0.0;
    colIndices = static_cast<int>(ceil(kwidthRow));
    for (int l{0}; l < colIndices; l++) {
      sumVal +=
          static_cast<double>(
              partialResize_data[((static_cast<int>(
                                       ipRowIndices_data[rowIdx + 227 * l]) +
                                   partialResize_size[0] * colIdx) +
                                  partialResize_size[0] * 227 * rowIndices) -
                                 1]) *
          (rowWeights_data[rowIdx + 227 * l] / rowWeightsTotal[rowIdx]);
    }
    sumVal = round(sumVal);
    if (sumVal < 256.0) {
      if (sumVal >= 0.0) {
        u = static_cast<unsigned char>(sumVal);
      } else {
        u = static_cast<unsigned char>(0U);
      }
    } else if (sumVal >= 256.0) {
      u = MAX_uint8_T;
    } else {
      u = static_cast<unsigned char>(0U);
    }
    out[(rowIdx + 227 * colIdx) + 51529 * rowIndices] = u;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double rowWeightsTotal[227]
//                const double rowWeights_data[2951]
//                const unsigned char img_data[2211840]
//                const int img_size[3]
//                const double ipRowIndices_data[2951]
//                const int partialResize_size[3]
//                const double kwidthRow
//                const int plast
//                unsigned char partialResize_data[697344]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void myAlexNetGPU_kernel19(
    const double rowWeightsTotal[227], const double rowWeights_data[2951],
    const unsigned char img_data[2211840], const int img_size[3],
    const double ipRowIndices_data[2951], const int partialResize_size[3],
    const double kwidthRow, const int plast,
    unsigned char partialResize_data[697344])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = 3UL * (227UL * (static_cast<unsigned long>(plast - 1) + 1UL)) - 1UL;
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    double sumVal;
    unsigned long tmpIndex;
    int colIdx;
    int colIndices;
    int rowIdx;
    int rowIndices;
    unsigned char u;
    rowIndices = static_cast<int>(idx % 3UL);
    tmpIndex = (idx - static_cast<unsigned long>(rowIndices)) / 3UL;
    rowIdx = static_cast<int>(tmpIndex % 227UL);
    tmpIndex = (tmpIndex - static_cast<unsigned long>(rowIdx)) / 227UL;
    colIdx = static_cast<int>(tmpIndex);
    sumVal = 0.0;
    colIndices = static_cast<int>(ceil(kwidthRow));
    for (int l{0}; l < colIndices; l++) {
      sumVal +=
          static_cast<double>(
              img_data[((static_cast<int>(ipRowIndices_data[rowIdx + 227 * l]) +
                         img_size[0] * colIdx) +
                        img_size[0] * img_size[1] * rowIndices) -
                       1]) *
          (rowWeights_data[rowIdx + 227 * l] / rowWeightsTotal[rowIdx]);
    }
    sumVal = round(sumVal);
    if (sumVal < 256.0) {
      if (sumVal >= 0.0) {
        u = static_cast<unsigned char>(sumVal);
      } else {
        u = static_cast<unsigned char>(0U);
      }
    } else if (sumVal >= 256.0) {
      u = MAX_uint8_T;
    } else {
      u = static_cast<unsigned char>(0U);
    }
    partialResize_data[(rowIdx + 227 * colIdx) +
                       227 * partialResize_size[1] * rowIndices] = u;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char I_data[]
//                const int I_size
//                unsigned char img_data[2211840]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void myAlexNetGPU_kernel2(
    const unsigned char I_data[], const int I_size,
    unsigned char img_data[2211840])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(I_size);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    img_data[oldIdx] = I_data[oldIdx];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double colWeightsTotal[227]
//                const double colWeights_data[4313]
//                const unsigned char partialResize_data[697344]
//                const int partialResize_size[3]
//                const double ipColIndices_data[4313]
//                const double kwidthCol
//                unsigned char out[154587]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void myAlexNetGPU_kernel20(
    const double colWeightsTotal[227], const double colWeights_data[4313],
    const unsigned char partialResize_data[697344],
    const int partialResize_size[3], const double ipColIndices_data[4313],
    const double kwidthCol, unsigned char out[154587])
{
  unsigned long threadId;
  int colIdx;
  int rowIdx;
  int rowIndices;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  rowIndices = static_cast<int>(threadId % 3UL);
  threadId = (threadId - static_cast<unsigned long>(rowIndices)) / 3UL;
  rowIdx = static_cast<int>(threadId % 227UL);
  threadId = (threadId - static_cast<unsigned long>(rowIdx)) / 227UL;
  colIdx = static_cast<int>(threadId);
  if ((static_cast<int>((static_cast<int>(colIdx < 227)) &&
                        (static_cast<int>(rowIdx < 227)))) &&
      (static_cast<int>(rowIndices < 3))) {
    double sumVal;
    int colIndices;
    unsigned char u;
    sumVal = 0.0;
    colIndices = static_cast<int>(ceil(kwidthCol));
    for (int l{0}; l < colIndices; l++) {
      sumVal +=
          static_cast<double>(
              partialResize_data
                  [(rowIdx + 227 * (static_cast<int>(
                                        ipColIndices_data[colIdx + 227 * l]) -
                                    1)) +
                   227 * partialResize_size[1] * rowIndices]) *
          (colWeights_data[colIdx + 227 * l] / colWeightsTotal[colIdx]);
    }
    sumVal = round(sumVal);
    if (sumVal < 256.0) {
      if (sumVal >= 0.0) {
        u = static_cast<unsigned char>(sumVal);
      } else {
        u = static_cast<unsigned char>(0U);
      }
    } else if (sumVal >= 256.0) {
      u = MAX_uint8_T;
    } else {
      u = static_cast<unsigned char>(0U);
    }
    out[(rowIdx + 227 * colIdx) + 51529 * rowIndices] = u;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char img_data[2211840]
//                const int img_size[3]
//                const int b_img_size[3]
//                const int plast
//                const int in_rows
//                unsigned char b_img_data[2211840]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void myAlexNetGPU_kernel3(
    const unsigned char img_data[2211840], const int img_size[3],
    const int b_img_size[3], const int plast, const int in_rows,
    unsigned char b_img_data[2211840])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<unsigned long>(plast) + 1UL) *
                ((static_cast<unsigned long>(in_rows) + 1UL) * 3UL) -
            1UL;
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    unsigned long tmpIndex;
    int colIndices;
    int oldIdx;
    int rowIndices;
    colIndices =
        static_cast<int>(idx % (static_cast<unsigned long>(plast) + 1UL));
    tmpIndex = (idx - static_cast<unsigned long>(colIndices)) /
               (static_cast<unsigned long>(plast) + 1UL);
    rowIndices = static_cast<int>(tmpIndex %
                                  (static_cast<unsigned long>(in_rows) + 1UL));
    tmpIndex = (tmpIndex - static_cast<unsigned long>(rowIndices)) /
               (static_cast<unsigned long>(in_rows) + 1UL);
    oldIdx = static_cast<int>(tmpIndex);
    b_img_data[(colIndices + b_img_size[0] * rowIndices) +
               b_img_size[0] * b_img_size[1] * oldIdx] =
        img_data[(colIndices + img_size[0] * rowIndices) +
                 img_size[0] * img_size[1] * (2 - oldIdx)];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char img_data[2211840]
//                const int img_size
//                unsigned char b_img_data[2211840]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void myAlexNetGPU_kernel4(
    const unsigned char img_data[2211840], const int img_size,
    unsigned char b_img_data[2211840])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(img_size);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    b_img_data[oldIdx] = img_data[oldIdx];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int in_rows
//                const int plast
//                double aux1_data[1440]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void myAlexNetGPU_kernel5(
    const int in_rows, const int plast, double aux1_data[1440])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(plast - 1);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int colIndices;
    colIndices = static_cast<int>(idx);
    if (colIndices + 1 <= in_rows) {
      aux1_data[colIndices] = static_cast<double>(colIndices) + 1.0;
    } else {
      aux1_data[colIndices] =
          (static_cast<double>(plast - colIndices) - 1.0) + 1.0;
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int in_cols
//                const int plast
//                double aux2_data[2048]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void myAlexNetGPU_kernel6(
    const int in_cols, const int plast, double aux2_data[2048])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(plast - 1);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int colIndices;
    colIndices = static_cast<int>(idx);
    if (colIndices + 1 <= in_cols) {
      aux2_data[colIndices] = static_cast<double>(colIndices) + 1.0;
    } else {
      aux2_data[colIndices] =
          (static_cast<double>(plast - colIndices) - 1.0) + 1.0;
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int plast
//                double ipRowIndices_data[2951]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void myAlexNetGPU_kernel7(
    const int plast, double ipRowIndices_data[2951])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(227 * plast - 1);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    ipRowIndices_data[oldIdx] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int plast
//                double rowWeights_data[2951]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void myAlexNetGPU_kernel8(
    const int plast, double rowWeights_data[2951])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(227 * plast - 1);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    rowWeights_data[oldIdx] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double aux1_data[1440]
//                const int in_rows
//                const double scale_idx_0
//                const double kwidthRow
//                double ipRowIndices_data[2951]
//                double rowWeights_data[2951]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void myAlexNetGPU_kernel9(
    const double aux1_data[1440], const int in_rows, const double scale_idx_0,
    const double kwidthRow, double ipRowIndices_data[2951],
    double rowWeights_data[2951])
{
  unsigned long threadId;
  int rowIdx;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  rowIdx = static_cast<int>(threadId);
  if (rowIdx < 227) {
    int colIndices;
    colIndices = static_cast<int>(ceil(kwidthRow));
    for (int k{0}; k < colIndices; k++) {
      double absx;
      double absx2;
      double sumVal;
      int b_y;
      int l;
      int rowIndices;
      sumVal = (static_cast<double>(rowIdx) + 1.0) / scale_idx_0 +
               0.5 * (1.0 - 1.0 / scale_idx_0);
      rowIndices = static_cast<int>(floor(sumVal - kwidthRow / 2.0));
      sumVal -= static_cast<double>(rowIndices + k) + 1.0;
      if (scale_idx_0 < 1.0) {
        sumVal *= scale_idx_0;
      }
      absx = fabs(sumVal);
      absx2 = absx * absx;
      sumVal = pow(absx, 3.0);
      sumVal = ((1.5 * sumVal - 2.5 * absx2) + 1.0) *
                   static_cast<double>(absx <= 1.0) +
               (((-0.5 * sumVal + 2.5 * absx2) - 4.0 * absx) + 2.0) *
                   static_cast<double>((static_cast<int>(1.0 < absx)) &&
                                       (static_cast<int>(absx <= 2.0)));
      if (scale_idx_0 < 1.0) {
        rowWeights_data[rowIdx + 227 * k] = scale_idx_0 * sumVal;
      } else {
        rowWeights_data[rowIdx + 227 * k] = sumVal;
      }
      b_y = in_rows << 1;
      rowIndices = (rowIndices + k) + 1;
      l = rowIndices - 1;
      if (b_y == 0) {
        if (rowIndices - 1 == 0) {
          l = 0;
        }
      } else if (rowIndices - 1 == 0) {
        l = 0;
      } else {
        l = static_cast<int>(fmod(static_cast<double>(rowIndices) - 1.0,
                                  static_cast<double>(b_y)));
        if ((static_cast<int>(l != 0)) &&
            (static_cast<int>(rowIndices - 1 < 0))) {
          l += b_y;
        }
      }
      ipRowIndices_data[rowIdx + 227 * k] = aux1_data[l];
    }
  }
}

//
// MYALEXNETGPU Accepts a 227x227x3 image to the deep neural network AlexNet
//  and returns the class index of the maximum confidence classification.
//
//  A list of all the classifications can be found in the file
//  classificationList.txt
//
//  Copyright 2018 The MathWorks, Inc.
//
// Arguments    : const unsigned char I_data[]
//                const int I_size[3]
// Return Type  : double
//
double myAlexNetGPU(const unsigned char I_data[], const int I_size[3])
{
  dim3 block;
  dim3 grid;
  double(*gpu_colWeights_data)[4313];
  double(*gpu_ipColIndices_data)[4313];
  double(*gpu_ipRowIndices_data)[2951];
  double(*gpu_rowWeights_data)[2951];
  double(*gpu_aux2_data)[2048];
  double(*gpu_aux1_data)[1440];
  double(*gpu_colWeightsTotal)[227];
  double(*gpu_rowWeightsTotal)[227];
  double classIdx;
  double kwidthCol;
  double kwidthRow;
  double scale_idx_0;
  double scale_idx_1;
  float output[1000];
  float ex;
  int b_img_size[3];
  int img_size[3];
  int partialResize_size[3];
  int(*b_gpu_img_size)[3];
  int(*gpu_I_size)[3];
  int(*gpu_img_size)[3];
  int(*gpu_partialResize_size)[3];
  int colWeights_size[2];
  int rowWeights_size[2];
  int in_cols;
  int plast;
  unsigned char(*b_gpu_img_data)[2211840];
  unsigned char(*gpu_img_data)[2211840];
  unsigned char(*b_gpu_partialResize_data)[697344];
  unsigned char(*gpu_partialResize_data)[490320];
  unsigned char out[154587];
  unsigned char(*gpu_out)[154587];
  unsigned char *gpu_I_data;
  bool b;
  bool validLaunchParams;
  if (!isInitialized_myAlexNetGPU) {
    myAlexNetGPU_initialize();
  }
  hipMalloc(&gpu_out, 154587UL);
  hipMalloc(&b_gpu_partialResize_data, 697344UL);
  hipMalloc(&gpu_partialResize_data, 490320UL);
  hipMalloc(&gpu_partialResize_size, 12UL);
  hipMalloc(&gpu_colWeightsTotal, 1816UL);
  hipMalloc(&gpu_rowWeightsTotal, 1816UL);
  hipMalloc(&gpu_colWeights_data, 34504UL);
  hipMalloc(&gpu_ipColIndices_data, 34504UL);
  hipMalloc(&gpu_rowWeights_data, 23608UL);
  hipMalloc(&gpu_ipRowIndices_data, 23608UL);
  hipMalloc(&gpu_aux2_data, 16384UL);
  hipMalloc(&gpu_aux1_data, 11520UL);
  hipMalloc(&b_gpu_img_data, 2211840UL);
  hipMalloc(&b_gpu_img_size, 12UL);
  hipMalloc(&gpu_img_size, 12UL);
  hipMalloc(&gpu_I_size, 12UL);
  hipMalloc(&gpu_img_data, 2211840UL);
  hipMalloc(&gpu_I_data, 2211840U * sizeof(unsigned char));
  //  Since the function "alexnet" is not supported for generation we load it
  //  from a MAT-file using coder.loadDeepLearningNetwork
  if (!net_not_empty) {
    coder::DeepLearningNetwork_setup(&net);
    net.matlabCodegenIsDeleted = false;
    net_not_empty = true;
  }
  //  Convert image data format OpenCV BGR to MATLAB compatible RGB image format
  //  Copyright 2020 The MathWorks, Inc.
  b = true;
  if ((I_size[1] != 0) && (I_size[2] != 0)) {
    bool exitg1;
    plast = 0;
    in_cols = 1;
    exitg1 = false;
    while ((!exitg1) && (in_cols < 4)) {
      if (I_size[3 - in_cols] != 1) {
        if (plast > 4 - in_cols) {
          b = false;
          exitg1 = true;
        } else {
          plast = 4 - in_cols;
          in_cols++;
        }
      } else {
        in_cols++;
      }
    }
  }
  if (b) {
    img_size[0] = I_size[2];
    img_size[1] = I_size[1];
    b = true;
    plast = I_size[1] * I_size[2] * 3 - 1;
    validLaunchParams = mwGetLaunchParameters1D(static_cast<double>(plast + 1L),
                                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      hipMemcpy(gpu_I_data, I_data,
                 3 * (I_size[1] * I_size[2]) * sizeof(unsigned char),
                 hipMemcpyHostToDevice);
      myAlexNetGPU_kernel2<<<grid, block>>>(gpu_I_data, plast, *gpu_img_data);
    }
  } else {
    img_size[0] = I_size[2];
    img_size[1] = I_size[1];
    b = true;
    in_cols = I_size[2];
    plast = I_size[1] - 1;
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<double>(3L *
                            (((I_size[1] - 1) + 1L) * ((I_size[2] - 1) + 1L))),
        &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      hipMemcpy(gpu_I_data, I_data,
                 3 * (I_size[1] * I_size[2]) * sizeof(unsigned char),
                 hipMemcpyHostToDevice);
      hipMemcpy(*gpu_I_size, I_size, 12UL, hipMemcpyHostToDevice);
      hipMemcpy(*gpu_img_size, img_size, 12UL, hipMemcpyHostToDevice);
      b = false;
      myAlexNetGPU_kernel1<<<grid, block>>>(gpu_I_data, *gpu_I_size,
                                            *gpu_img_size, plast, in_cols,
                                            *gpu_img_data);
    }
  }
  plast = img_size[0] - 1;
  in_cols = img_size[1] - 1;
  b_img_size[0] = img_size[0];
  b_img_size[1] = img_size[1];
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<double>(((img_size[0] - 1) + 1L) *
                          (((img_size[1] - 1) + 1L) * 3L)),
      &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    if (b) {
      hipMemcpy(*gpu_img_size, img_size, 12UL, hipMemcpyHostToDevice);
    }
    hipMemcpy(*b_gpu_img_size, b_img_size, 12UL, hipMemcpyHostToDevice);
    myAlexNetGPU_kernel3<<<grid, block>>>(*gpu_img_data, *gpu_img_size,
                                          *b_gpu_img_size, plast, in_cols,
                                          *b_gpu_img_data);
  }
  img_size[0] = b_img_size[0];
  img_size[1] = b_img_size[1];
  plast = b_img_size[0] * b_img_size[1] * 3 - 1;
  validLaunchParams = mwGetLaunchParameters1D(static_cast<double>(plast + 1L),
                                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    myAlexNetGPU_kernel4<<<grid, block>>>(*b_gpu_img_data, plast,
                                          *gpu_img_data);
  }
  // sz = size(img);
  // sizeWH = sz([2 1]);
  //  Resize
  scale_idx_0 = 227.0 / static_cast<double>(b_img_size[0]);
  scale_idx_1 = 227.0 / static_cast<double>(b_img_size[1]);
  plast = b_img_size[0] << 1;
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<double>((plast - 1) + 1L), &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    myAlexNetGPU_kernel5<<<grid, block>>>(b_img_size[0], plast, *gpu_aux1_data);
  }
  plast = b_img_size[1] << 1;
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<double>((plast - 1) + 1L), &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    myAlexNetGPU_kernel6<<<grid, block>>>(b_img_size[1], plast, *gpu_aux2_data);
  }
  if (scale_idx_0 < 1.0) {
    kwidthRow = 4.0 / scale_idx_0;
  } else {
    kwidthRow = 4.0;
  }
  plast = static_cast<int>(std::ceil(kwidthRow));
  validLaunchParams =
      mwGetLaunchParameters1D(static_cast<double>((227 * plast - 1) + 1L),
                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    myAlexNetGPU_kernel7<<<grid, block>>>(plast, *gpu_ipRowIndices_data);
  }
  plast = static_cast<int>(std::ceil(kwidthRow));
  rowWeights_size[1] = plast;
  validLaunchParams =
      mwGetLaunchParameters1D(static_cast<double>((227 * plast - 1) + 1L),
                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    myAlexNetGPU_kernel8<<<grid, block>>>(plast, *gpu_rowWeights_data);
  }
  myAlexNetGPU_kernel9<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
      *gpu_aux1_data, b_img_size[0], scale_idx_0, kwidthRow,
      *gpu_ipRowIndices_data, *gpu_rowWeights_data);
  if (scale_idx_1 < 1.0) {
    kwidthCol = 4.0 / scale_idx_1;
  } else {
    kwidthCol = 4.0;
  }
  plast = static_cast<int>(std::ceil(kwidthCol));
  validLaunchParams =
      mwGetLaunchParameters1D(static_cast<double>((227 * plast - 1) + 1L),
                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    myAlexNetGPU_kernel10<<<grid, block>>>(plast, *gpu_ipColIndices_data);
  }
  plast = static_cast<int>(std::ceil(kwidthCol));
  colWeights_size[1] = plast;
  validLaunchParams =
      mwGetLaunchParameters1D(static_cast<double>((227 * plast - 1) + 1L),
                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    myAlexNetGPU_kernel11<<<grid, block>>>(plast, *gpu_colWeights_data);
  }
  myAlexNetGPU_kernel12<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
      *gpu_aux2_data, b_img_size[1], scale_idx_1, kwidthCol,
      *gpu_ipColIndices_data, *gpu_colWeights_data);
  plast = rowWeights_size[1];
  myAlexNetGPU_kernel13<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
      *gpu_rowWeights_data, *gpu_rowWeightsTotal);
  for (in_cols = 0; in_cols <= plast - 2; in_cols++) {
    myAlexNetGPU_kernel14<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        *gpu_rowWeights_data, (in_cols + 1) * 227, *gpu_rowWeightsTotal);
  }
  plast = colWeights_size[1];
  myAlexNetGPU_kernel15<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
      *gpu_colWeights_data, *gpu_colWeightsTotal);
  for (in_cols = 0; in_cols <= plast - 2; in_cols++) {
    myAlexNetGPU_kernel16<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        *gpu_colWeights_data, (in_cols + 1) * 227, *gpu_colWeightsTotal);
  }
  if (!(scale_idx_0 > scale_idx_1)) {
    partialResize_size[1] = b_img_size[1];
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<double>(3L * (227L * ((b_img_size[1] - 1) + 1L))), &grid,
        &block, 1024U, 65535U);
    if (validLaunchParams) {
      hipMemcpy(*gpu_img_size, img_size, 12UL, hipMemcpyHostToDevice);
      hipMemcpy(*gpu_partialResize_size, partialResize_size, 12UL,
                 hipMemcpyHostToDevice);
      myAlexNetGPU_kernel19<<<grid, block>>>(
          *gpu_rowWeightsTotal, *gpu_rowWeights_data, *gpu_img_data,
          *gpu_img_size, *gpu_ipRowIndices_data, *gpu_partialResize_size,
          kwidthRow, b_img_size[1], *b_gpu_partialResize_data);
    } else {
      hipMemcpy(*gpu_partialResize_size, partialResize_size, 12UL,
                 hipMemcpyHostToDevice);
    }
    myAlexNetGPU_kernel20<<<dim3(302U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_colWeightsTotal, *gpu_colWeights_data, *b_gpu_partialResize_data,
        *gpu_partialResize_size, *gpu_ipColIndices_data, kwidthCol, *gpu_out);
  } else {
    partialResize_size[0] = b_img_size[0];
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<double>(3L * (227L * ((b_img_size[0] - 1) + 1L))), &grid,
        &block, 1024U, 65535U);
    if (validLaunchParams) {
      hipMemcpy(*gpu_img_size, img_size, 12UL, hipMemcpyHostToDevice);
      hipMemcpy(*gpu_partialResize_size, partialResize_size, 12UL,
                 hipMemcpyHostToDevice);
      myAlexNetGPU_kernel17<<<grid, block>>>(
          *gpu_colWeightsTotal, *gpu_colWeights_data, *gpu_img_data,
          *gpu_ipColIndices_data, *gpu_img_size, *gpu_partialResize_size,
          kwidthCol, b_img_size[0], *gpu_partialResize_data);
    } else {
      hipMemcpy(*gpu_partialResize_size, partialResize_size, 12UL,
                 hipMemcpyHostToDevice);
    }
    myAlexNetGPU_kernel18<<<dim3(302U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_rowWeightsTotal, *gpu_rowWeights_data, *gpu_partialResize_data,
        *gpu_partialResize_size, *gpu_ipRowIndices_data, kwidthRow, *gpu_out);
  }
  //  Predict with AlexNet
  hipMemcpy(out, *gpu_out, 154587UL, hipMemcpyDeviceToHost);
  coder::DeepLearningNetwork_predict(&net, out, output);
  //  Determine the class index with the highest probability
  plast = 1;
  ex = output[0];
  for (in_cols = 0; in_cols < 999; in_cols++) {
    float f;
    f = output[in_cols + 1];
    if (std::isnan(f)) {
      b = false;
    } else if (std::isnan(ex)) {
      b = true;
    } else {
      b = (ex < f);
    }
    if (b) {
      ex = f;
      plast = in_cols + 2;
    }
  }
  classIdx = plast;
  hipFree(gpu_I_data);
  hipFree(*gpu_img_data);
  hipFree(*gpu_I_size);
  hipFree(*gpu_img_size);
  hipFree(*b_gpu_img_size);
  hipFree(*b_gpu_img_data);
  hipFree(*gpu_aux1_data);
  hipFree(*gpu_aux2_data);
  hipFree(*gpu_ipRowIndices_data);
  hipFree(*gpu_rowWeights_data);
  hipFree(*gpu_ipColIndices_data);
  hipFree(*gpu_colWeights_data);
  hipFree(*gpu_rowWeightsTotal);
  hipFree(*gpu_colWeightsTotal);
  hipFree(*gpu_partialResize_size);
  hipFree(*gpu_partialResize_data);
  hipFree(*b_gpu_partialResize_data);
  hipFree(*gpu_out);
  return classIdx;
}

//
// MYALEXNETGPU Accepts a 227x227x3 image to the deep neural network AlexNet
//  and returns the class index of the maximum confidence classification.
//
//  A list of all the classifications can be found in the file
//  classificationList.txt
//
//  Copyright 2018 The MathWorks, Inc.
//
// Arguments    : void
// Return Type  : void
//
void myAlexNetGPU_free()
{
  if (!net.matlabCodegenIsDeleted) {
    net.matlabCodegenIsDeleted = true;
    coder::DeepLearningNetwork_delete(&net);
  }
}

//
// MYALEXNETGPU Accepts a 227x227x3 image to the deep neural network AlexNet
//  and returns the class index of the maximum confidence classification.
//
//  A list of all the classifications can be found in the file
//  classificationList.txt
//
//  Copyright 2018 The MathWorks, Inc.
//
// Arguments    : void
// Return Type  : void
//
void myAlexNetGPU_init()
{
  net_not_empty = false;
  net.matlabCodegenIsDeleted = true;
}

//
// File trailer for myAlexNetGPU.cu
//
// [EOF]
//
