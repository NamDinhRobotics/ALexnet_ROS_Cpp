#include "MWCudnnTargetNetworkImpl.hpp"
#include "MWTargetNetworkImplBase.hpp"
#include "MWTargetTypes.hpp"
#include "MWCudnnCNNLayerImpl.hpp"
#include "MWCudnnLayerImplFactory.hpp"
#include <cassert>
#include <cmath>
#include <algorithm>
 namespace MWCudnnTarget { MWTargetNetworkImpl::MWTargetNetworkImpl() : 
MWTargetNetworkImplBase(MWTargetType::CUDNN_TARGET, new 
MWCudnnLayerImplFactory) , xcusoQxPPodcHwVviCWI(0) , 
NZjOkZPwLzQsdEVkwMcX(0) , NbunkIVaMPVYgAQHXXYd(0) , MW_autoTune(true) 
, leWFtIPrKkXLixGWBGJW(0) , GsZlHFuhbvjLtRMDjXnW(0) {} 
void MWTargetNetworkImpl::allocate(int BufSize, int numBufsToAlloc) { numBufs = 
numBufsToAlloc; memBuffer.reserve(numBufs); for(int i = 0; i < numBufs; i++) { 
float *memPtr = 0; CUDA_CALL(hipMalloc((void**)&memPtr, 
sizeof(float)*BufSize)); memBuffer.push_back(memPtr); }  } void 
MWTargetNetworkImpl::allocatePermuteBuffers(int bufSize, int numBufsToAlloc) { 
for (int i = 0; i < numBufsToAlloc; i++) { float* memPtr = 0; 
CUDA_CALL(hipMalloc((void**)&memPtr, sizeof(float) * bufSize)); 
kqftrrQBBOgGsrDSkIUk.push_back(memPtr); } } void 
MWTargetNetworkImpl::preSetup() {  NZjOkZPwLzQsdEVkwMcX = new 
hipblasHandle_t; if(!NZjOkZPwLzQsdEVkwMcX) { 
MWCNNLayerImpl::throwAllocationError(__LINE__ , __FILE__); } 
CUBLAS_CALL(hipblasCreate(NZjOkZPwLzQsdEVkwMcX)); NbunkIVaMPVYgAQHXXYd 
= new hipdnnHandle_t; if(!NbunkIVaMPVYgAQHXXYd) { 
MWCNNLayerImpl::throwAllocationError(__LINE__ , __FILE__); } 
CUDNN_CALL(hipdnnCreate(NbunkIVaMPVYgAQHXXYd));  } void 
MWTargetNetworkImpl::postSetup(MWCNNLayer* layers[],int numLayers) { if 
(*getProposedWorkSpaceSize() > *getAllocatedWorkSpaceSize()) { if 
(xcusoQxPPodcHwVviCWI) { destroyWorkSpace(xcusoQxPPodcHwVviCWI); } 
createWorkSpace(xcusoQxPPodcHwVviCWI); while ((!xcusoQxPPodcHwVviCWI) && 
(*getProposedWorkSpaceSize() > 0)) { 
setProposedWorkSpaceSize(MWTargetNetworkImpl::getNextProposedWorkSpaceSize(*getProposedWorkSpaceSize())); 
createWorkSpace(xcusoQxPPodcHwVviCWI); } } for (int i = 0; i < numLayers; i++) 
{ layers[i]->postSetup();  }  } size_t 
MWTargetNetworkImpl::getNextProposedWorkSpaceSize(size_t failedWorkSpaceSize) { 
assert(failedWorkSpaceSize > 0); return failedWorkSpaceSize/2; } void 
MWTargetNetworkImpl::createWorkSpace(float*& xkUNToJIgvoLoUQuzKRF) { 
hipError_t rMMjgjGRAiLVlTlRSByU = hipMalloc((void**)&xkUNToJIgvoLoUQuzKRF, 
*getProposedWorkSpaceSize()); if (rMMjgjGRAiLVlTlRSByU != hipSuccess) { 
xkUNToJIgvoLoUQuzKRF = NULL; setAllocatedWorkSpaceSize(0);  
rMMjgjGRAiLVlTlRSByU = hipGetLastError();  } else { 
setAllocatedWorkSpaceSize(*getProposedWorkSpaceSize()); } } void 
MWTargetNetworkImpl::destroyWorkSpace(float*& xkUNToJIgvoLoUQuzKRF) { 
CUDA_FREE_CALL(xkUNToJIgvoLoUQuzKRF); xkUNToJIgvoLoUQuzKRF = NULL; 
setAllocatedWorkSpaceSize(0);  } void 
MWTargetNetworkImpl::setProposedWorkSpaceSize(size_t wss) { 
leWFtIPrKkXLixGWBGJW = wss;  } size_t* 
MWTargetNetworkImpl::getProposedWorkSpaceSize() { return 
&leWFtIPrKkXLixGWBGJW; } void 
MWTargetNetworkImpl::setAllocatedWorkSpaceSize(size_t wss) { 
GsZlHFuhbvjLtRMDjXnW = wss;  } size_t* 
MWTargetNetworkImpl::getAllocatedWorkSpaceSize() { return 
&GsZlHFuhbvjLtRMDjXnW; } float* 
MWTargetNetworkImpl::getWorkSpace() { return xcusoQxPPodcHwVviCWI; } float* 
MWTargetNetworkImpl::getPermuteBuffer(int bufIndex) { return 
kqftrrQBBOgGsrDSkIUk[bufIndex]; } hipblasHandle_t* 
MWTargetNetworkImpl::getCublasHandle() { return NZjOkZPwLzQsdEVkwMcX; } 
hipdnnHandle_t* MWTargetNetworkImpl::getCudnnHandle() { return 
NbunkIVaMPVYgAQHXXYd; } void MWTargetNetworkImpl::setAutoTune(bool 
autotune) { MW_autoTune = autotune; } bool MWTargetNetworkImpl::getAutoTune() 
const { return MW_autoTune; } void MWTargetNetworkImpl::deallocate() { for(int 
i = 0; i < memBuffer.size(); i++) { float *memPtr = memBuffer[i]; if(memPtr) { 
CUDA_FREE_CALL(memPtr); }  } memBuffer.clear(); for(int i = 0; i < 
kqftrrQBBOgGsrDSkIUk.size(); i++) { float *memPtr = 
kqftrrQBBOgGsrDSkIUk[i]; if(memPtr) { CUDA_FREE_CALL(memPtr); } } 
kqftrrQBBOgGsrDSkIUk.clear(); } void MWTargetNetworkImpl::cleanup() { if 
(xcusoQxPPodcHwVviCWI) { destroyWorkSpace(xcusoQxPPodcHwVviCWI); } if 
(NZjOkZPwLzQsdEVkwMcX) { hipError_t hipError_t = hipPeekAtLastError(); if 
(hipError_t != hipErrorDeinitialized) { 
CUBLAS_CALL(hipblasDestroy(*NZjOkZPwLzQsdEVkwMcX)); } delete 
NZjOkZPwLzQsdEVkwMcX; } if (NbunkIVaMPVYgAQHXXYd) {
#if (CUDNN_MAJOR < 8) 
 CUDNN_CALL(hipdnnDestroy(*NbunkIVaMPVYgAQHXXYd));
#else
 hipError_t hipError_t = hipPeekAtLastError();  if (hipError_t != 
hipErrorDeinitialized) { 
CUDNN_CALL(hipdnnDestroy(*NbunkIVaMPVYgAQHXXYd)); }
#endif
 delete NbunkIVaMPVYgAQHXXYd; } } float* 
MWTargetNetworkImpl::getBufferPtr(int bufferIndex) { 
assert(static_cast<size_t>(bufferIndex) < memBuffer.size()); return 
memBuffer[bufferIndex]; } } 