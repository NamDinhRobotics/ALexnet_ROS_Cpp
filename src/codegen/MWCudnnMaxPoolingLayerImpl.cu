#include "hip/hip_runtime.h"
#include "MWCudnnMaxPoolingLayerImpl.hpp"
#include "MWCudnnCNNLayerImpl.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWCNNLayer.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
 namespace MWCudnnTarget { void __global__ MWSetDyForBackPropImpl(float * 
OwortPcLToImGdYFtbSF, const int eqOmMKQRpqBqRQCnJmxt); void __global__ 
doMWMaxPoolingLayerImpl(float * URgvgDXnZskIYGdtimcU, float * 
UKtMXCCqdjeyaVHabkxg, const int BRSPqxNffoBYKqpSVHne); 
MWMaxPoolingLayerImpl::MWMaxPoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int DGzdAcREJHGXjyRzNjJV,  int 
ECTnqgWHyHCHCLBZlffd,  int FOcStuqCptsGIZXskVpC,  int 
FpguQZSermqZCMRiUfML, int CTCbzQMDaLxINPbODdng, int 
CLOUhPjbgggWoXHTtmjC,  int CpMjJjtGOeWOzwxpAAQP, int 
CqtPRJvHlGJFssiPzsOm, bool GDRXdUDklKFEYEfifhIH,  int eqUIJyhXTwRqtPfXapcx) 
: MWCNNLayerImpl(layer, ntwk_impl) , 
BLjrjqvCcCommiXWQLjs(GDRXdUDklKFEYEfifhIH) , URgvgDXnZskIYGdtimcU(0) 
, OwortPcLToImGdYFtbSF(0) , DCdZnqpcBnvXVgEsLBnz(DGzdAcREJHGXjyRzNjJV) , 
DqxLTLaJwwgQqmrtCDuu(ECTnqgWHyHCHCLBZlffd) , 
CufLFODQDXTAPyRqYodN(DGzdAcREJHGXjyRzNjJV) , 
DSsxcjIrUgZCKZovyNQf(ECTnqgWHyHCHCLBZlffd) , 
CGbFsczkgkhjcHoCKzBx(CTCbzQMDaLxINPbODdng) , 
CDJtexcMbXMWAmnNZsNf(CLOUhPjbgggWoXHTtmjC) , 
CZNYmBcNFSZWvaCklqeM(CpMjJjtGOeWOzwxpAAQP) , 
ClEhcJFlvGCgiavziIag(CqtPRJvHlGJFssiPzsOm) , 
FLuSVNoPhAFKtLUchSvv(FOcStuqCptsGIZXskVpC) , 
FeVcBgtQmTLtmnNcJGMY(FpguQZSermqZCMRiUfML) , 
puSFZkRJmyuFPfQRswDK(std::pow(2, layer->getScalingExponent())) , 
vFNECEAeLZsYsUxvlgqL(std::string{"INT8x4"}.compare(layer->getAccelMode()) 
== 0) , etjQLJVQCaeAXRWYtqOl(eqUIJyhXTwRqtPfXapcx) {  
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&lHtftnmGBvlSSoGOXVui)); 
createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex()); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&DRzwhbNPpftRRIXXfHzd));  } 
MWMaxPoolingLayerImpl::~MWMaxPoolingLayerImpl() { } void 
MWMaxPoolingLayerImpl::propagateSize() {  MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor(0); MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); if ((DCdZnqpcBnvXVgEsLBnz == -1) && 
(DqxLTLaJwwgQqmrtCDuu == -1)) { CufLFODQDXTAPyRqYodN = 
ipTensorBase->getHeight(); DSsxcjIrUgZCKZovyNQf = ipTensorBase->getWidth(); } int 
lWJYwWaFPmWNQDPrlqER = CGbFsczkgkhjcHoCKzBx; int 
lXJKIOEATumoVKStGbVy = CZNYmBcNFSZWvaCklqeM; 
CUDNN_CALL(hipdnnSetPooling2dDescriptor(lHtftnmGBvlSSoGOXVui, HIPDNN_POOLING_MAX, 
HIPDNN_NOT_PROPAGATE_NAN, CufLFODQDXTAPyRqYodN, DSsxcjIrUgZCKZovyNQf, 
lWJYwWaFPmWNQDPrlqER, lXJKIOEATumoVKStGbVy, FLuSVNoPhAFKtLUchSvv, 
FeVcBgtQmTLtmnNcJGMY)); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensorBase->getSourcePortIndex()); assert(desc); if 
(ipTensorBase->isInt8()) { if (vFNECEAeLZsYsUxvlgqL) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(DRzwhbNPpftRRIXXfHzd, 
HIPDNN_TENSOR_NCHW_VECT_C,  HIPDNN_DATA_INT8x4,  ipTensorBase->getBatchSize(),  
ipTensorBase->getChannels(),  ipTensorBase->getHeight(),  
ipTensorBase->getWidth())  ); MWCNNLayerImpl::setDescriptorForINT8(*desc, 
static_cast<MWTensor<signed char>*>(opTensorBase), HIPDNN_DATA_INT8x4, 
HIPDNN_TENSOR_NCHW_VECT_C);  } else { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(DRzwhbNPpftRRIXXfHzd, 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_INT8, ipTensorBase->getBatchSize(), 
ipTensorBase->getChannels(), ipTensorBase->getHeight(), 
ipTensorBase->getWidth())); MWCNNLayerImpl::setDescriptorForINT8(*desc, 
static_cast<MWTensor<signed char>*>(opTensorBase), HIPDNN_DATA_INT8, 
HIPDNN_TENSOR_NCHW); } } else { setDescriptor<float>(*desc, 
static_cast<MWTensor<float>*>(opTensorBase)); } } void 
MWMaxPoolingLayerImpl::allocate() { MWCNNLayer* maxpoolLayer = getLayer(); 
MWTensorBase* ipTensor = maxpoolLayer->getInputTensor(0); MWTensorBase* 
opTensor = maxpoolLayer->getOutputTensor(0); if (BLjrjqvCcCommiXWQLjs){ const 
int dJcdBfQQLhIAYHPxwQeg = ipTensor->getNumElements(); 
CUDA_CALL(hipMalloc((void**)&URgvgDXnZskIYGdtimcU, 
sizeof(float)*dJcdBfQQLhIAYHPxwQeg)); const int eqOmMKQRpqBqRQCnJmxt = 
opTensor->getNumElements(); CUDA_CALL(hipMalloc((void**)&OwortPcLToImGdYFtbSF, 
sizeof(float)*eqOmMKQRpqBqRQCnJmxt)); int shEncNmxJsMuJKwbrwok = 
(eqOmMKQRpqBqRQCnJmxt < 1024) ? eqOmMKQRpqBqRQCnJmxt : 1024; int 
KHClOltUSuqFVVErSxVb = (eqOmMKQRpqBqRQCnJmxt + shEncNmxJsMuJKwbrwok - 
1)/shEncNmxJsMuJKwbrwok; 
MWSetDyForBackPropImpl<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( OwortPcLToImGdYFtbSF, eqOmMKQRpqBqRQCnJmxt); } } void 
MWMaxPoolingLayerImpl::deallocate() { if (URgvgDXnZskIYGdtimcU){ 
CUDA_FREE_CALL(URgvgDXnZskIYGdtimcU); URgvgDXnZskIYGdtimcU = 
NULL; } if (OwortPcLToImGdYFtbSF){ CUDA_FREE_CALL(OwortPcLToImGdYFtbSF); OwortPcLToImGdYFtbSF = 
NULL; }  } void MWMaxPoolingLayerImpl::predict() { MWCNNLayer* maxpoolLayer = 
getLayer(); MWTensorBase* ipTensorBase = maxpoolLayer->getInputTensor(0); 
MWTensorBase* opTensorBase = maxpoolLayer->getOutputTensor(0); 
hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensorBase->getSourcePortIndex()); assert(desc); 
hipdnnTensorDescriptor_t XYbzSmRQGatVJtGmDZSo; if (opTensorBase->isInt8()) { 
XYbzSmRQGatVJtGmDZSo = DRzwhbNPpftRRIXXfHzd; MWTensor<signed char>* ipTensor = 
static_cast<MWTensor<signed char>*>(ipTensorBase); MWTensor<signed char>* 
opTensor = static_cast<MWTensor<signed char>*>(opTensorBase); 
CUDNN_CALL(hipdnnPoolingForward(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
lHtftnmGBvlSSoGOXVui, &puSFZkRJmyuFPfQRswDK, XYbzSmRQGatVJtGmDZSo, 
ipTensor->getData(), getZeroPtr(), *desc, opTensor->getData())); } else { 
XYbzSmRQGatVJtGmDZSo = MWCNNLayerImpl::getCuDNNDescriptor(ipTensorBase); 
MWTensor<float>* ipTensor = static_cast<MWTensor<float>*>(ipTensorBase); 
MWTensor<float>* opTensor = static_cast<MWTensor<float>*>(opTensorBase); 
CUDNN_CALL(hipdnnPoolingForward(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
lHtftnmGBvlSSoGOXVui, getOnePtr(), XYbzSmRQGatVJtGmDZSo, ipTensor->getData(), 
getZeroPtr(), *desc, opTensor->getData())); if (BLjrjqvCcCommiXWQLjs) { 
CUDNN_CALL(hipdnnPoolingBackward(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
lHtftnmGBvlSSoGOXVui, getOnePtr(), *desc, opTensor->getData(), *desc, 
OwortPcLToImGdYFtbSF, XYbzSmRQGatVJtGmDZSo, ipTensor->getData(), getZeroPtr(), 
XYbzSmRQGatVJtGmDZSo, URgvgDXnZskIYGdtimcU)); int dJcdBfQQLhIAYHPxwQeg = 
ipTensor->getNumElements(); int shEncNmxJsMuJKwbrwok = 
(dJcdBfQQLhIAYHPxwQeg < 1024) ? dJcdBfQQLhIAYHPxwQeg : 1024; int 
KHClOltUSuqFVVErSxVb = (dJcdBfQQLhIAYHPxwQeg + shEncNmxJsMuJKwbrwok - 
1)/shEncNmxJsMuJKwbrwok; 
doMWMaxPoolingLayerImpl<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( URgvgDXnZskIYGdtimcU, 
static_cast<MWTensor<float>*>(maxpoolLayer->getOutputTensor(1))->getData(), 
dJcdBfQQLhIAYHPxwQeg); }  } return; } void MWMaxPoolingLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(lHtftnmGBvlSSoGOXVui)); MWTensorBase* 
opTensorBase = getLayer()->getOutputTensor(0);  if (opTensorBase->isInt8()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(DRzwhbNPpftRRIXXfHzd)); } } float* 
MWMaxPoolingLayerImpl::getIndexData()  { return 
static_cast<MWTensor<float>*>(getLayer()->getOutputTensor(1))->getData(); } 
void __global__ __launch_bounds__(1024) MWSetDyForBackPropImpl(float * 
OwortPcLToImGdYFtbSF, const int eqOmMKQRpqBqRQCnJmxt) { for(int i = blockDim.x * 
blockIdx.x + threadIdx.x; i < eqOmMKQRpqBqRQCnJmxt; i+= blockDim.x*gridDim.x) { 
OwortPcLToImGdYFtbSF[i] = i+1; } } void __global__ __launch_bounds__(1024) 
doMWMaxPoolingLayerImpl(float * URgvgDXnZskIYGdtimcU, float * 
UKtMXCCqdjeyaVHabkxg, const int BRSPqxNffoBYKqpSVHne) { for(int i = blockDim.x * 
blockIdx.x + threadIdx.x; i < BRSPqxNffoBYKqpSVHne; i+= blockDim.x*gridDim.x) { if 
(static_cast<int>(URgvgDXnZskIYGdtimcU[i]) != 0){ 
UKtMXCCqdjeyaVHabkxg[static_cast<int>(URgvgDXnZskIYGdtimcU[i])-1] = 
i; } } } } 