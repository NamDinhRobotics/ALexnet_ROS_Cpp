#include "MWCudnnFusedConvActivationLayerImpl.hpp"
#include "MWCudnnCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
#include <cassert>
#include <cstdio>
#include <vector>
 namespace MWCudnnTarget { 
MWFusedConvActivationLayerImpl::MWFusedConvActivationLayerImpl(MWCNNLayer* 
layer, MWTargetNetworkImpl* ntwk_impl, int filt_H, int filt_W, int numGrps, int 
numChnls, int numFilts, int FOcStuqCptsGIZXskVpC, int 
FpguQZSermqZCMRiUfML, int CTCbzQMDaLxINPbODdng, int 
CLOUhPjbgggWoXHTtmjC, int CpMjJjtGOeWOzwxpAAQP, int 
CqtPRJvHlGJFssiPzsOm, int AjhVZuQXURJimwbnYqDF, int 
AwZQzUhuWVLGrWgLHRuM, int edQOkUJIZbwzEeIcCLzG, const char* 
xHViLEwTujGGrPZZgmbF, const char* JxwPQNPACGfmGpNncpCY, double , 
MWActivationFunctionType::ACTIVATION_FCN_ENUM GZGFVDrXwFLJleoTDywO) : 
MWCNNLayerImpl(layer, ntwk_impl) , vIWQzNvYZSuxmOTVDFhU(NULL) , IwKnaBoXVubIRYcxEJLH(NULL) , 
TfsmDFpPPOscKZifVzSQ(NULL) , WprSrhAStKGxyXeoxETy(NULL) , 
FshVHIJMRAhtQirYPlZd(NULL) , HgeIbZCtKXtKFOEtSlPZ(NULL) , 
AzTsxYcYjIEJsGQbeYHm(filt_H) , BHuHNDGoRwGRouCxeMbw (filt_W) , 
BkwhtPQUCQKchmmimoXs (numGrps) , BUOdotSvmFyUWQKMUdra (numChnls) , 
BdqURaHPmdnfzvtUvocl (numFilts) , FLuSVNoPhAFKtLUchSvv(FOcStuqCptsGIZXskVpC) 
, FeVcBgtQmTLtmnNcJGMY(FpguQZSermqZCMRiUfML) , 
CGbFsczkgkhjcHoCKzBx(CTCbzQMDaLxINPbODdng) , 
CDJtexcMbXMWAmnNZsNf(CLOUhPjbgggWoXHTtmjC) , 
CZNYmBcNFSZWvaCklqeM(CpMjJjtGOeWOzwxpAAQP) , 
ClEhcJFlvGCgiavziIag(CqtPRJvHlGJFssiPzsOm) , 
AdmgfUbRAfzFeYHxSnQr(AjhVZuQXURJimwbnYqDF) , 
AuqaQHxmPQSyYRemQvyX(AwZQzUhuWVLGrWgLHRuM) , 
fSKMHAqIghbYYgyIpNDw(edQOkUJIZbwzEeIcCLzG) , 
IAlDgIFcchbwRGBSfVfA((CGbFsczkgkhjcHoCKzBx != CDJtexcMbXMWAmnNZsNf) 
|| (CZNYmBcNFSZWvaCklqeM != ClEhcJFlvGCgiavziIag)) { 
assert(GZGFVDrXwFLJleoTDywO == MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU);
#if (CUDNN_MAJOR < 6)
 throw std::runtime_error("Fused ConvReLU Layer only supported for cuDNN 6 or greater");
#else
 cQBKlCKXxecGPJrXBXdk = ntwk_impl; 
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&NMMfJylfQjiIUAKhXCJb)); 
CUDNN_CALL(hipdnnCreateFilterDescriptor(&PtkeOkuClHzhOfpmBevf)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&JsZenQeBPMhwsyEhVHiD));  
CUDNN_CALL(hipdnnCreateActivationDescriptor(&muwRQxtWMMXAPxSuMYBw)); 
MWTensorBase* ipTensor_conv = getLayer()->getInputTensor(0); int 
NNhshzQGJHLSGjDiVerE = CGbFsczkgkhjcHoCKzBx; int 
NXruhrCCiguRjAgSNDuz = CZNYmBcNFSZWvaCklqeM; if 
(IAlDgIFcchbwRGBSfVfA) { NNhshzQGJHLSGjDiVerE = 0; 
NXruhrCCiguRjAgSNDuz = 0; TfsmDFpPPOscKZifVzSQ = new MWTensor<float>(-1, 
-1, -1, -1, -1, NULL, getLayer(), ipTensor_conv->getDataFormat(), 0); if 
(!TfsmDFpPPOscKZifVzSQ) { MWCNNLayerImpl::throwAllocationError(__LINE__ , 
__FILE__); } CUDNN_CALL(hipdnnCreateTensorDescriptor(&XYbzSmRQGatVJtGmDZSo)); } 
else { TfsmDFpPPOscKZifVzSQ = ipTensor_conv; } assert(TfsmDFpPPOscKZifVzSQ != 
NULL); bERCRkGjpaKXMNComoYl = CGbFsczkgkhjcHoCKzBx; bOrQjJTNlssnrexxbHdi = 
CZNYmBcNFSZWvaCklqeM; 
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(NMMfJylfQjiIUAKhXCJb, 
NNhshzQGJHLSGjDiVerE, NXruhrCCiguRjAgSNDuz, FLuSVNoPhAFKtLUchSvv, 
FeVcBgtQmTLtmnNcJGMY, AdmgfUbRAfzFeYHxSnQr, AuqaQHxmPQSyYRemQvyX, 
HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
#if (FP16_ENABLED == 1 && ( CUDNN_MAJOR > 7 || (CUDNN_MAJOR == 7 && CUDNN_MINOR >= 2) ))
 CUDNN_CALL(hipdnnSetConvolutionMathType(NMMfJylfQjiIUAKhXCJb, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
#endif
 if (BkwhtPQUCQKchmmimoXs > 1){ 
CUDNN_CALL(hipdnnSetConvolutionGroupCount(NMMfJylfQjiIUAKhXCJb, 
BkwhtPQUCQKchmmimoXs)); } 
CUDNN_CALL(hipdnnSetActivationDescriptor(muwRQxtWMMXAPxSuMYBw, 
HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0)); int 
eWYFXrUazhqiEIscccda = BUOdotSvmFyUWQKMUdra*BkwhtPQUCQKchmmimoXs; int 
eqmVWbEcwBRGnVNDUtrG = BdqURaHPmdnfzvtUvocl*BkwhtPQUCQKchmmimoXs; 
CUDNN_CALL(hipdnnSetFilter4dDescriptor(PtkeOkuClHzhOfpmBevf, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, eqmVWbEcwBRGnVNDUtrG, 
eWYFXrUazhqiEIscccda/BkwhtPQUCQKchmmimoXs, AzTsxYcYjIEJsGQbeYHm, 
BHuHNDGoRwGRouCxeMbw)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(JsZenQeBPMhwsyEhVHiD, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, eqmVWbEcwBRGnVNDUtrG, 1, 1)); int weightSize = 
BUOdotSvmFyUWQKMUdra*eqmVWbEcwBRGnVNDUtrG*AzTsxYcYjIEJsGQbeYHm*BHuHNDGoRwGRouCxeMbw; 
CUDA_CALL(hipMalloc((void**)&vIWQzNvYZSuxmOTVDFhU, sizeof(float)*weightSize)); 
CUDA_CALL(hipMalloc((void**)&IwKnaBoXVubIRYcxEJLH, 
sizeof(float)*eqmVWbEcwBRGnVNDUtrG)); 
loadWeights(xHViLEwTujGGrPZZgmbF); loadBias(JxwPQNPACGfmGpNncpCY); createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex());
#endif
 } MWFusedConvActivationLayerImpl::~MWFusedConvActivationLayerImpl() { } void 
MWFusedConvActivationLayerImpl::propagateSize() {
#if (CUDNN_MAJOR >= 6)
 MWTensorBase* ipTensor_conv = getLayer()->getInputTensor(0); int inputH; int 
inputW; if (IAlDgIFcchbwRGBSfVfA) { inputH = 
ipTensor_conv->getHeight() + CGbFsczkgkhjcHoCKzBx + CDJtexcMbXMWAmnNZsNf; 
inputW = ipTensor_conv->getWidth() + CZNYmBcNFSZWvaCklqeM + 
ClEhcJFlvGCgiavziIag; } else { inputH = ipTensor_conv->getHeight(); inputW = 
ipTensor_conv->getWidth(); } TfsmDFpPPOscKZifVzSQ->setHeight(inputH); 
TfsmDFpPPOscKZifVzSQ->setWidth(inputW); 
TfsmDFpPPOscKZifVzSQ->setChannels(ipTensor_conv->getChannels()); 
TfsmDFpPPOscKZifVzSQ->setBatchSize(ipTensor_conv->getBatchSize()); 
TfsmDFpPPOscKZifVzSQ->setSequenceLength(ipTensor_conv->getSequenceLength()); 
assert(TfsmDFpPPOscKZifVzSQ->getSequenceLength() == 1); if 
(IAlDgIFcchbwRGBSfVfA) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(XYbzSmRQGatVJtGmDZSo, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, TfsmDFpPPOscKZifVzSQ->getBatchSize(), 
TfsmDFpPPOscKZifVzSQ->getChannels(), TfsmDFpPPOscKZifVzSQ->getHeight(), 
TfsmDFpPPOscKZifVzSQ->getWidth())); } else { XYbzSmRQGatVJtGmDZSo = 
MWCNNLayerImpl::getCuDNNDescriptor(TfsmDFpPPOscKZifVzSQ); } 
assert(BUOdotSvmFyUWQKMUdra == 
TfsmDFpPPOscKZifVzSQ->getChannels()/BkwhtPQUCQKchmmimoXs); MWTensorBase* opTensor 
= getLayer()->getOutputTensor(0); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor));
#if (CUDNN_MAJOR < 7)
 { 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, PtkeOkuClHzhOfpmBevf, NMMfJylfQjiIUAKhXCJb, *desc, 
HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &NDjzAZSYJuWymuKDNZYB)); }
#else
 { const int maxAlgoCount(3); int returnedAlgoCount(-1); 
hipdnnConvolutionFwdAlgoPerf_t perf_results[maxAlgoCount]; 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, PtkeOkuClHzhOfpmBevf, NMMfJylfQjiIUAKhXCJb, *desc, 
maxAlgoCount, &returnedAlgoCount, perf_results)); NDjzAZSYJuWymuKDNZYB = 
perf_results[0].algo; }
#endif
 if (CUDNN_VERSION < 7402) fixConvAlgo(); size_t sxuOMwKXOKfuExclRaSe = 0; 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, PtkeOkuClHzhOfpmBevf, NMMfJylfQjiIUAKhXCJb, *desc, 
NDjzAZSYJuWymuKDNZYB, &sxuOMwKXOKfuExclRaSe)); if( sxuOMwKXOKfuExclRaSe > 
*cQBKlCKXxecGPJrXBXdk->getProposedWorkSpaceSize() ) { 
cQBKlCKXxecGPJrXBXdk->setProposedWorkSpaceSize(sxuOMwKXOKfuExclRaSe); }
#endif
 } void MWFusedConvActivationLayerImpl::allocate() { MWTensorBase* 
ipTensor_conv = getLayer()->getInputTensor(0); if 
(IAlDgIFcchbwRGBSfVfA) { float* newInput; int inputH = 
ipTensor_conv->getHeight() + CGbFsczkgkhjcHoCKzBx + CDJtexcMbXMWAmnNZsNf; int 
inputW = ipTensor_conv->getWidth() + CZNYmBcNFSZWvaCklqeM + 
ClEhcJFlvGCgiavziIag; int paddedSize = ipTensor_conv->getBatchSize() * 
ipTensor_conv->getChannels() * inputH * inputW; 
CUDA_CALL(hipMalloc((void**)&newInput, sizeof(float)*paddedSize)); 
CUDA_CALL(hipMemset(newInput, 0, sizeof(float)*paddedSize)); 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->setData(newInput); } 
WprSrhAStKGxyXeoxETy = 
static_cast<MWTensor<float>*>(getLayer()->getOutputTensor(0))->getData(); 
setalpha2Ptr(getZeroPtr()); int numInputs = getLayer()->getNumInputs(); if 
(numInputs == 2) { setalpha2Ptr(getOnePtr()); WprSrhAStKGxyXeoxETy = 
static_cast<MWTensor<float>*>(getLayer()->getInputTensor(1))->getData(); } if 
(static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData() == 
WprSrhAStKGxyXeoxETy){ int xInputTensorSize = 
getLayer()->getInputTensor(0)->getNumElements(); 
CUDA_CALL(hipMalloc((void**)&FshVHIJMRAhtQirYPlZd, sizeof(float) * 
xInputTensorSize)); } } void MWFusedConvActivationLayerImpl::deallocate() { if 
(TfsmDFpPPOscKZifVzSQ != getLayer()->getInputTensor(0)) { 
assert(IAlDgIFcchbwRGBSfVfA); 
CUDA_FREE_CALL(static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData()); 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->setData((float*)NULL); } if 
(FshVHIJMRAhtQirYPlZd){ CUDA_FREE_CALL(FshVHIJMRAhtQirYPlZd); 
FshVHIJMRAhtQirYPlZd = NULL;  }  } void 
MWFusedConvActivationLayerImpl::predict() { MWCNNLayer* fusedConvReluLayer = 
getLayer(); MWTensorBase* ipTensorBase = fusedConvReluLayer->getInputTensor(); 
MWTensorBase* opTensorBase = fusedConvReluLayer->getOutputTensor(); 
MWTensor<float>* ipTensor = static_cast<MWTensor<float>*>(ipTensorBase); 
MWTensor<float>* opTensor = static_cast<MWTensor<float>*>(opTensorBase); if 
(TfsmDFpPPOscKZifVzSQ != fusedConvReluLayer->getInputTensor()) { 
CUDA_CALL(hipMemset(static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(), 
0, sizeof(float)*TfsmDFpPPOscKZifVzSQ->getNumElements())); 
MWCNNLayerImpl::padInput(ipTensor->getData(), ipTensor->getHeight(), 
ipTensor->getWidth(), ipTensor->getChannels(), TfsmDFpPPOscKZifVzSQ->getHeight(), 
TfsmDFpPPOscKZifVzSQ->getWidth(), bERCRkGjpaKXMNComoYl, bOrQjJTNlssnrexxbHdi, 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(), 
ipTensor->getNumElements()); } hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc);
#if (CUDNN_MAJOR >= 6)
 assert(opTensor->getData() != 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData() || 
(getLayer()->getNumInputs() == 2)); float* pzUAoBDvaKAtdsmkQuct; if 
(static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData() == 
WprSrhAStKGxyXeoxETy){ CUDA_CALL(hipMemcpy(FshVHIJMRAhtQirYPlZd, 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(), sizeof(float) * 
opTensorBase->getNumElements(), hipMemcpyDeviceToDevice)); 
pzUAoBDvaKAtdsmkQuct = FshVHIJMRAhtQirYPlZd; }else{ 
pzUAoBDvaKAtdsmkQuct = 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(); } 
CUDNN_CALL(cudnnConvolutionBiasActivationForward(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
getOnePtr(), XYbzSmRQGatVJtGmDZSo, pzUAoBDvaKAtdsmkQuct, 
PtkeOkuClHzhOfpmBevf, vIWQzNvYZSuxmOTVDFhU, NMMfJylfQjiIUAKhXCJb, NDjzAZSYJuWymuKDNZYB, 
cQBKlCKXxecGPJrXBXdk->getWorkSpace(), 
*cQBKlCKXxecGPJrXBXdk->getAllocatedWorkSpaceSize(), getalpha2Ptr(),  *desc,  
WprSrhAStKGxyXeoxETy,  JsZenQeBPMhwsyEhVHiD, IwKnaBoXVubIRYcxEJLH, muwRQxtWMMXAPxSuMYBw, 
*desc, opTensor->getData()));
#endif
 } void MWFusedConvActivationLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(NMMfJylfQjiIUAKhXCJb)); 
CUDNN_CALL(hipdnnDestroyFilterDescriptor(PtkeOkuClHzhOfpmBevf)); 
CUDNN_CALL(hipdnnDestroyActivationDescriptor(muwRQxtWMMXAPxSuMYBw)); if 
(vIWQzNvYZSuxmOTVDFhU) { CUDA_FREE_CALL(vIWQzNvYZSuxmOTVDFhU); vIWQzNvYZSuxmOTVDFhU = NULL; } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(JsZenQeBPMhwsyEhVHiD)); if 
(IwKnaBoXVubIRYcxEJLH) { CUDA_FREE_CALL(IwKnaBoXVubIRYcxEJLH); IwKnaBoXVubIRYcxEJLH = NULL; } if 
(TfsmDFpPPOscKZifVzSQ != getLayer()->getInputTensor(0)) { 
assert(IAlDgIFcchbwRGBSfVfA); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(XYbzSmRQGatVJtGmDZSo)); } } void 
MWFusedConvActivationLayerImpl::loadWeights(const char* PmFfARVzoHVAYkfpuvqK) { 
FILE* QMgBqCuvjnbWHWiVPEwn = MWCNNLayer::openBinaryFile(PmFfARVzoHVAYkfpuvqK); 
assert(QMgBqCuvjnbWHWiVPEwn); int cRtIUoZRPICuQEOZOSzT = 
BUOdotSvmFyUWQKMUdra*BkwhtPQUCQKchmmimoXs*BdqURaHPmdnfzvtUvocl*AzTsxYcYjIEJsGQbeYHm*BHuHNDGoRwGRouCxeMbw; 
 float* KZWeXiYFmdpQdsgidKeG = MALLOC_CALL(sizeof(float)*cRtIUoZRPICuQEOZOSzT); 
MWCNNLayer::call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), cRtIUoZRPICuQEOZOSzT, 
QMgBqCuvjnbWHWiVPEwn, PmFfARVzoHVAYkfpuvqK); CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, 
KZWeXiYFmdpQdsgidKeG, sizeof(float)*cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice));
#if 0
 printf("%s loaded. Size = %d. %f\n", PmFfARVzoHVAYkfpuvqK, cRtIUoZRPICuQEOZOSzT, KZWeXiYFmdpQdsgidKeG[0]);
#endif
 free(KZWeXiYFmdpQdsgidKeG); fclose(QMgBqCuvjnbWHWiVPEwn); return; } void 
MWFusedConvActivationLayerImpl::loadBias(const char* PmFfARVzoHVAYkfpuvqK) { 
FILE* QMgBqCuvjnbWHWiVPEwn = MWCNNLayer::openBinaryFile(PmFfARVzoHVAYkfpuvqK); 
assert(QMgBqCuvjnbWHWiVPEwn); int cRtIUoZRPICuQEOZOSzT = 
BkwhtPQUCQKchmmimoXs*BdqURaHPmdnfzvtUvocl;  float* KZWeXiYFmdpQdsgidKeG = 
MALLOC_CALL(sizeof(float)*cRtIUoZRPICuQEOZOSzT); 
MWCNNLayer::call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), cRtIUoZRPICuQEOZOSzT, 
QMgBqCuvjnbWHWiVPEwn, PmFfARVzoHVAYkfpuvqK); CUDA_CALL(hipMemcpy(IwKnaBoXVubIRYcxEJLH, 
KZWeXiYFmdpQdsgidKeG, sizeof(float)*cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); 
free(KZWeXiYFmdpQdsgidKeG); fclose(QMgBqCuvjnbWHWiVPEwn); return; } void 
MWFusedConvActivationLayerImpl::setLearnables(std::vector<float*> learnables) { 
assert(learnables.size() == 2); int cRtIUoZRPICuQEOZOSzT = BUOdotSvmFyUWQKMUdra * 
BdqURaHPmdnfzvtUvocl * BkwhtPQUCQKchmmimoXs * AzTsxYcYjIEJsGQbeYHm * 
BHuHNDGoRwGRouCxeMbw;  float* wqggPBXZvtlxnxwngvAq = learnables[0]; 
CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, wqggPBXZvtlxnxwngvAq, sizeof(float) * 
cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); cRtIUoZRPICuQEOZOSzT = 
BdqURaHPmdnfzvtUvocl * BkwhtPQUCQKchmmimoXs;  float* JgLfgHrHMEMmMYTettJF = 
learnables[1]; CUDA_CALL(hipMemcpy(IwKnaBoXVubIRYcxEJLH, JgLfgHrHMEMmMYTettJF, 
sizeof(float) * cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); } void 
MWFusedConvActivationLayerImpl::postSetup() { if 
(cQBKlCKXxecGPJrXBXdk->getAutoTune()) { getConvAlgoTuned(); } else { 
getConvAlgoWorkSpaceLimit(); } } void 
MWFusedConvActivationLayerImpl::getConvAlgoTuned() { MWTensorBase* opTensorBase 
= getLayer()->getOutputTensor(0); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); hipdnnConvolutionFwdAlgoPerf_t 
perf_results[3]; hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); 
assert(desc); int returnedAlgoCount; 
CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithmEx(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(), 
PtkeOkuClHzhOfpmBevf, vIWQzNvYZSuxmOTVDFhU, NMMfJylfQjiIUAKhXCJb, *desc, 
opTensor->getData(), 3, &returnedAlgoCount, &perf_results[0], 
cQBKlCKXxecGPJrXBXdk->getWorkSpace(), 
*cQBKlCKXxecGPJrXBXdk->getAllocatedWorkSpaceSize())); NDjzAZSYJuWymuKDNZYB = 
perf_results[0].algo; if (CUDNN_VERSION < 7402) fixConvAlgo(); } void 
MWFusedConvActivationLayerImpl::getConvAlgoWorkSpaceLimit() { 
hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); assert(desc);
#if (CUDNN_MAJOR < 8)
 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, PtkeOkuClHzhOfpmBevf, NMMfJylfQjiIUAKhXCJb, *desc, 
HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT, 
*cQBKlCKXxecGPJrXBXdk->getAllocatedWorkSpaceSize(), &NDjzAZSYJuWymuKDNZYB));
#else
 int maxAlgoCount(-1); 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithmMaxCount(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
&maxAlgoCount)); int returnedAlgoCount(-1); 
std::vector<hipdnnConvolutionFwdAlgoPerf_t> perf_results(maxAlgoCount);  
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, PtkeOkuClHzhOfpmBevf, NMMfJylfQjiIUAKhXCJb, *desc, 
maxAlgoCount, &returnedAlgoCount, &perf_results[0])); 
hipdnnConvolutionFwdAlgoPerf_t nextFastest; bool algoFound(false); for (int i = 
0; i < returnedAlgoCount; ++i) { nextFastest = perf_results[i]; if 
(nextFastest.memory <= *cQBKlCKXxecGPJrXBXdk->getAllocatedWorkSpaceSize()) { 
NDjzAZSYJuWymuKDNZYB = nextFastest.algo; algoFound = true; break; } } assert(algoFound);
#endif
 if (CUDNN_VERSION < 7402) fixConvAlgo(); } void 
MWFusedConvActivationLayerImpl::fixConvAlgo() { int inputH = 
TfsmDFpPPOscKZifVzSQ->getHeight(); int inputW = TfsmDFpPPOscKZifVzSQ->getWidth(); 
if (NDjzAZSYJuWymuKDNZYB == HIPDNN_CONVOLUTION_FWD_ALGO_FFT && (inputH > 64 || 
inputW > 64)) { NDjzAZSYJuWymuKDNZYB = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM; 
} } } 