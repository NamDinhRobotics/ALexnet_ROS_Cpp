#include "MWCudnnFCLayerImpl.hpp"
#include "MWCudnnCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
#include <vector>
 namespace MWCudnnTarget { MWFCLayerImpl::MWFCLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int XLJXOFXdnZOyJvtltbyr, int 
kMyEnepVyoNObTPqIpWo, const char* xHViLEwTujGGrPZZgmbF, const char* 
JxwPQNPACGfmGpNncpCY) : MWCNNLayerImpl(layer, ntwk_impl) , 
BlRIQPyqJZORKENzSdYf(XLJXOFXdnZOyJvtltbyr) , 
BuyZFXzwOMxcePIbCLfl(kMyEnepVyoNObTPqIpWo) , vIWQzNvYZSuxmOTVDFhU(NULL) , 
vpXxoeEhdEosLSsYXkNG(NULL) , IwKnaBoXVubIRYcxEJLH(NULL) , 
xHiBGayUfxIpXKkCTDNU(false) { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&JsZenQeBPMhwsyEhVHiD)); 
createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex()); 
CUDA_CALL(hipMalloc((void**)&vIWQzNvYZSuxmOTVDFhU, sizeof(float) * 
BlRIQPyqJZORKENzSdYf * BuyZFXzwOMxcePIbCLfl)); 
CUDA_CALL(hipMalloc((void**)&IwKnaBoXVubIRYcxEJLH, sizeof(float) * 
BuyZFXzwOMxcePIbCLfl)); vpXxoeEhdEosLSsYXkNG = 
MALLOC_CALL(sizeof(float) * BlRIQPyqJZORKENzSdYf * 
BuyZFXzwOMxcePIbCLfl); loadWeights(xHViLEwTujGGrPZZgmbF); 
loadBias(JxwPQNPACGfmGpNncpCY); } MWFCLayerImpl::~MWFCLayerImpl() { } void 
MWFCLayerImpl::propagateSize() { MWCNNLayer* fcLayer = getLayer(); 
MWTensorBase* opTensor = fcLayer->getOutputTensor(0); hipdnnTensorDescriptor_t* 
desc = getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor)); if 
(opTensor->getSequenceLength() == 1) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(JsZenQeBPMhwsyEhVHiD, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, BuyZFXzwOMxcePIbCLfl, 1, 1)); } else { int dims[5] 
= {1, 1, BuyZFXzwOMxcePIbCLfl, 1, 1}; int strides[5]; 
MWTensorBase::getStrides(dims, 5, strides); CUDNN_CALL( 
hipdnnSetTensorNdDescriptor(JsZenQeBPMhwsyEhVHiD, HIPDNN_DATA_FLOAT, 5, dims, 
strides)); } } void MWFCLayerImpl::loadWeights(const char* PmFfARVzoHVAYkfpuvqK) 
{ FILE* QMgBqCuvjnbWHWiVPEwn = MWCNNLayer::openBinaryFile(PmFfARVzoHVAYkfpuvqK); 
assert(QMgBqCuvjnbWHWiVPEwn); int cRtIUoZRPICuQEOZOSzT = BlRIQPyqJZORKENzSdYf * 
BuyZFXzwOMxcePIbCLfl;  MWCNNLayer::call_fread(vpXxoeEhdEosLSsYXkNG, 
sizeof(float), cRtIUoZRPICuQEOZOSzT, QMgBqCuvjnbWHWiVPEwn, PmFfARVzoHVAYkfpuvqK); 
fclose(QMgBqCuvjnbWHWiVPEwn); } void MWFCLayerImpl::prepareWeights(float* 
wqggPBXZvtlxnxwngvAq) { int cRtIUoZRPICuQEOZOSzT = BlRIQPyqJZORKENzSdYf * 
BuyZFXzwOMxcePIbCLfl; MWCNNLayer* fcLayer = getLayer(); MWTensorBase* 
ipTensor = fcLayer->getInputTensor(0); if (ipTensor->getHeight() != 1 && 
ipTensor->getWidth() != 1) { float* KZWeXiYFmdpQdsgidKeG = 
MALLOC_CALL(sizeof(float) * ipTensor->getHeight() * ipTensor->getWidth()); for 
(int k = 0; k < cRtIUoZRPICuQEOZOSzT / ipTensor->getHeight() / ipTensor->getWidth(); 
k++) { for (int i = 0; i < ipTensor->getHeight() * ipTensor->getWidth(); i++) 
KZWeXiYFmdpQdsgidKeG[i] = wqggPBXZvtlxnxwngvAq[k * ipTensor->getHeight() * 
ipTensor->getWidth() + i]; for (int j = 0; j < ipTensor->getHeight(); j++) for 
(int i = 0; i < ipTensor->getWidth(); i++) wqggPBXZvtlxnxwngvAq[k * 
ipTensor->getHeight() * ipTensor->getWidth() + j * ipTensor->getWidth() + i] = 
KZWeXiYFmdpQdsgidKeG[j + i * ipTensor->getHeight()]; } free(KZWeXiYFmdpQdsgidKeG); } 
CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, wqggPBXZvtlxnxwngvAq, sizeof(float) * 
cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); } void MWFCLayerImpl::loadBias(const 
char* PmFfARVzoHVAYkfpuvqK) { MWCNNLayer* fcLayer = getLayer(); MWTensorBase* 
opTensor = fcLayer->getOutputTensor(0); FILE* QMgBqCuvjnbWHWiVPEwn = 
MWCNNLayer::openBinaryFile(PmFfARVzoHVAYkfpuvqK); assert(QMgBqCuvjnbWHWiVPEwn); int 
cRtIUoZRPICuQEOZOSzT = BuyZFXzwOMxcePIbCLfl;  float* KZWeXiYFmdpQdsgidKeG = 
MALLOC_CALL(sizeof(float) * cRtIUoZRPICuQEOZOSzT); 
MWCNNLayer::call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), cRtIUoZRPICuQEOZOSzT, 
QMgBqCuvjnbWHWiVPEwn, PmFfARVzoHVAYkfpuvqK); CUDA_CALL(hipMemcpy(IwKnaBoXVubIRYcxEJLH, 
KZWeXiYFmdpQdsgidKeG, sizeof(float) * cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); 
free(KZWeXiYFmdpQdsgidKeG); fclose(QMgBqCuvjnbWHWiVPEwn); } void 
MWFCLayerImpl::setLearnables(std::vector<float*> learnables) { 
assert(learnables.size() == 2);  float* wqggPBXZvtlxnxwngvAq = learnables[0]; 
prepareWeights(wqggPBXZvtlxnxwngvAq); float* JgLfgHrHMEMmMYTettJF = learnables[1]; 
CUDA_CALL(hipMemcpy(IwKnaBoXVubIRYcxEJLH, JgLfgHrHMEMmMYTettJF, sizeof(float) * 
BuyZFXzwOMxcePIbCLfl, hipMemcpyHostToDevice)); } void 
MWFCLayerImpl::postSetup() { if (!xHiBGayUfxIpXKkCTDNU) { 
prepareWeights(vpXxoeEhdEosLSsYXkNG); free(vpXxoeEhdEosLSsYXkNG); 
vpXxoeEhdEosLSsYXkNG = NULL; xHiBGayUfxIpXKkCTDNU = true; } } void 
MWFCLayerImpl::predict() { MWCNNLayer* fcLayer = getLayer(); MWTensorBase* 
ipTensorBase = fcLayer->getInputTensor(0); MWTensorBase* opTensorBase = 
fcLayer->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); int numOutputRows = 
opTensor->getChannels(); int numOutputCols = ipTensor->getBatchSize() * 
ipTensor->getSequenceLength(); int innerDimension = ipTensor->getHeight() * 
ipTensor->getWidth() * ipTensor->getChannels(); int TxNFOfYScyqGlEFFxbAv = 1; int 
UEESbUvbMihFnquvuFij = 1; if (opTensor->getBatchSize() == 1 && 
opTensor->getSequenceLength() == 1) { CUDA_CALL(hipMemcpy(opTensor->getData(), 
IwKnaBoXVubIRYcxEJLH, sizeof(float) * numOutputRows, hipMemcpyDeviceToDevice)); 
CUBLAS_CALL(hipblasSgemv(*cQBKlCKXxecGPJrXBXdk->getCublasHandle(), HIPBLAS_OP_T, 
innerDimension, numOutputRows, getOnePtr(), vIWQzNvYZSuxmOTVDFhU, innerDimension, 
ipTensor->getData(), TxNFOfYScyqGlEFFxbAv, getOnePtr(), opTensor->getData(), 
UEESbUvbMihFnquvuFij)); } else { 
CUBLAS_CALL(hipblasSgemm(*cQBKlCKXxecGPJrXBXdk->getCublasHandle(), HIPBLAS_OP_T, 
HIPBLAS_OP_N, numOutputRows, numOutputCols, innerDimension, getOnePtr(), 
vIWQzNvYZSuxmOTVDFhU, innerDimension, ipTensor->getData(), innerDimension, 
getZeroPtr(), opTensor->getData(), numOutputRows)); hipdnnTensorDescriptor_t* 
desc = getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
CUDNN_CALL(hipdnnAddTensor(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), getOnePtr(), 
JsZenQeBPMhwsyEhVHiD, IwKnaBoXVubIRYcxEJLH, getOnePtr(), *desc, opTensor->getData())); } 
return; } void MWFCLayerImpl::cleanup() { if (vIWQzNvYZSuxmOTVDFhU) { 
CUDA_FREE_CALL(vIWQzNvYZSuxmOTVDFhU); vIWQzNvYZSuxmOTVDFhU = NULL; } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(JsZenQeBPMhwsyEhVHiD)); if 
(IwKnaBoXVubIRYcxEJLH) { CUDA_FREE_CALL(IwKnaBoXVubIRYcxEJLH); IwKnaBoXVubIRYcxEJLH = NULL; } } } 