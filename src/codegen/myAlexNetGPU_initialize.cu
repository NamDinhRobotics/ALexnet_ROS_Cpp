//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: myAlexNetGPU_initialize.cu
//
// GPU Coder version                    : 2.2
// CUDA/C/C++ source code generated on  : 10-Dec-2021 14:26:27
//

// Include Files
#include "myAlexNetGPU_initialize.h"
#include "myAlexNetGPU.h"
#include "myAlexNetGPU_data.h"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void myAlexNetGPU_initialize()
{
  myAlexNetGPU_init();
  hipGetLastError();
  isInitialized_myAlexNetGPU = true;
}

//
// File trailer for myAlexNetGPU_initialize.cu
//
// [EOF]
//
