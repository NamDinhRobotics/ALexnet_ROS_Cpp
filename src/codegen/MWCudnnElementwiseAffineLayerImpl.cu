#include "MWCudnnElementwiseAffineLayerImpl.hpp"
#include "MWCudnnCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
#include "MWKernelHeaders.hpp"
#include <cmath>
#include <cassert>
#include <cstdio>
 namespace MWCudnnTarget { 
MWElementwiseAffineLayerImpl::MWElementwiseAffineLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int scale_H, int scale_W, int scale_C, int 
offset_H, int offset_W, int offset_C, bool isClipped, int lowerbound, int 
upperbound, const char* pdleXafalaHAmketaFyq, const char* 
gsJtSpgIkTNvahoTFqow) : MWCNNLayerImpl(layer, ntwk_impl), 
olKGEIcsxmLSoMhRhEtP(NULL), fYaOQTeunPwVjnhhTECh(NULL), pFoPPXxxFRbjXXxQWItv(scale_H), 
pckLLTEdVPoCZLRwyDnM(scale_W), osBZbKVTgXwTSsGSbdth(scale_C), 
gTcJMwtYuwiqqUmqvKhT(offset_H), gcGbhKACQPAogUYXHedj(offset_W), 
gNROjwaqhxDPvBWUCUcQ(offset_C), ZDWLzHUkuZuIUZHfbGDY(isClipped), 
bDTIjtxZiSHtjwzgEluE(lowerbound), unSXtdjDjpysqxmbIiPv(upperbound) { 
CUDA_CALL(hipMalloc((void**)&olKGEIcsxmLSoMhRhEtP, 
sizeof(float)*pFoPPXxxFRbjXXxQWItv*pckLLTEdVPoCZLRwyDnM*osBZbKVTgXwTSsGSbdth)); 
CUDA_CALL(hipMalloc((void**)&fYaOQTeunPwVjnhhTECh, 
sizeof(float)*gTcJMwtYuwiqqUmqvKhT*gcGbhKACQPAogUYXHedj*gNROjwaqhxDPvBWUCUcQ));  
loadScale(pdleXafalaHAmketaFyq); loadOffset(gsJtSpgIkTNvahoTFqow); } 
MWElementwiseAffineLayerImpl::~MWElementwiseAffineLayerImpl() { } void 
MWElementwiseAffineLayerImpl::propagateSize() { } void 
MWElementwiseAffineLayerImpl::predict() { MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor(0); MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); int WerBmCOBWhvoFbdqfitc = 
ipTensor->getHeight(); int WmXADZOqdcQvtBUvFerh = ipTensor->getWidth(); int 
WOJynDmqVUPWjAGVIuMQ = ipTensor->getChannels(); long int 
YNDVziqpDddiXQKYZZhX = WerBmCOBWhvoFbdqfitc*WmXADZOqdcQvtBUvFerh; long 
int YMNbgnUYZspjMLjwcIOS = 
YNDVziqpDddiXQKYZZhX*WOJynDmqVUPWjAGVIuMQ; long int 
YGiQICncmsGZkNUyiQyg = ipTensor->getNumElements(); long int rlQsibXJSWJVnUVpdNeL = 
((YGiQICncmsGZkNUyiQyg + 31) / 32) * 32; int shEncNmxJsMuJKwbrwok = 
(rlQsibXJSWJVnUVpdNeL < 1024) ? rlQsibXJSWJVnUVpdNeL : 1024; long int 
KHClOltUSuqFVVErSxVb = (YGiQICncmsGZkNUyiQyg + shEncNmxJsMuJKwbrwok - 
1) / shEncNmxJsMuJKwbrwok; long int pbePKOGQbvmzToFbiRkR = 
pFoPPXxxFRbjXXxQWItv * pckLLTEdVPoCZLRwyDnM * osBZbKVTgXwTSsGSbdth; long int 
gWETwFdWHfKuelmlKNCC = gTcJMwtYuwiqqUmqvKhT * gcGbhKACQPAogUYXHedj * 
gNROjwaqhxDPvBWUCUcQ; assert(pbePKOGQbvmzToFbiRkR <= YGiQICncmsGZkNUyiQyg); 
assert(gWETwFdWHfKuelmlKNCC <= YGiQICncmsGZkNUyiQyg); if (pbePKOGQbvmzToFbiRkR == 
1) { scale_scalar_kernel<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( ipTensor->getData(),  opTensor->getData(), 
olKGEIcsxmLSoMhRhEtP, YGiQICncmsGZkNUyiQyg); } else if (pFoPPXxxFRbjXXxQWItv == 1 && 
pckLLTEdVPoCZLRwyDnM == 1 && pbePKOGQbvmzToFbiRkR > 1) { 
scale_vector_kernel<<<KHClOltUSuqFVVErSxVb, shEncNmxJsMuJKwbrwok>>>( 
ipTensor->getData(),  opTensor->getData(), olKGEIcsxmLSoMhRhEtP, 
YNDVziqpDddiXQKYZZhX, YMNbgnUYZspjMLjwcIOS, 
YGiQICncmsGZkNUyiQyg); } else if (YMNbgnUYZspjMLjwcIOS == 
pbePKOGQbvmzToFbiRkR) {  scale_tensor3d_kernel<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( ipTensor->getData(),  opTensor->getData(), 
olKGEIcsxmLSoMhRhEtP,  YMNbgnUYZspjMLjwcIOS, YGiQICncmsGZkNUyiQyg); } else 
{ scale_matrix2d_kernel<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( ipTensor->getData(),  opTensor->getData(), 
olKGEIcsxmLSoMhRhEtP,  YNDVziqpDddiXQKYZZhX, YGiQICncmsGZkNUyiQyg); } if 
(gWETwFdWHfKuelmlKNCC == 1) { offset_scalar_kernel<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( opTensor->getData(),  opTensor->getData(), 
fYaOQTeunPwVjnhhTECh, YGiQICncmsGZkNUyiQyg, ZDWLzHUkuZuIUZHfbGDY, 
bDTIjtxZiSHtjwzgEluE, unSXtdjDjpysqxmbIiPv); } else if (gTcJMwtYuwiqqUmqvKhT 
== 1 && gcGbhKACQPAogUYXHedj == 1 && gWETwFdWHfKuelmlKNCC > 1) { 
offset_vector_kernel<<<KHClOltUSuqFVVErSxVb, shEncNmxJsMuJKwbrwok>>>( 
opTensor->getData(),  opTensor->getData(), fYaOQTeunPwVjnhhTECh, 
YNDVziqpDddiXQKYZZhX, YMNbgnUYZspjMLjwcIOS, 
YGiQICncmsGZkNUyiQyg, ZDWLzHUkuZuIUZHfbGDY, bDTIjtxZiSHtjwzgEluE, 
unSXtdjDjpysqxmbIiPv); } else if (YMNbgnUYZspjMLjwcIOS == 
gWETwFdWHfKuelmlKNCC) { offset_tensor3d_kernel<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( opTensor->getData(),  opTensor->getData(), 
fYaOQTeunPwVjnhhTECh, YMNbgnUYZspjMLjwcIOS, YGiQICncmsGZkNUyiQyg, 
ZDWLzHUkuZuIUZHfbGDY, bDTIjtxZiSHtjwzgEluE, unSXtdjDjpysqxmbIiPv); } else { 
offset_matrix2d_kernel<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( opTensor->getData(),  opTensor->getData(), 
fYaOQTeunPwVjnhhTECh, YNDVziqpDddiXQKYZZhX, YGiQICncmsGZkNUyiQyg, 
ZDWLzHUkuZuIUZHfbGDY, bDTIjtxZiSHtjwzgEluE, unSXtdjDjpysqxmbIiPv); } return; 
} void MWElementwiseAffineLayerImpl::cleanup() { if (olKGEIcsxmLSoMhRhEtP) { 
CUDA_FREE_CALL(olKGEIcsxmLSoMhRhEtP); olKGEIcsxmLSoMhRhEtP = NULL; } if 
(fYaOQTeunPwVjnhhTECh) { CUDA_FREE_CALL(fYaOQTeunPwVjnhhTECh); fYaOQTeunPwVjnhhTECh = 
NULL; }  } void MWElementwiseAffineLayerImpl::loadScale(const char* 
pdleXafalaHAmketaFyq) { FILE* QMgBqCuvjnbWHWiVPEwn = 
MWCNNLayer::openBinaryFile(pdleXafalaHAmketaFyq); assert(QMgBqCuvjnbWHWiVPEwn); long 
int cRtIUoZRPICuQEOZOSzT = pFoPPXxxFRbjXXxQWItv*pckLLTEdVPoCZLRwyDnM*osBZbKVTgXwTSsGSbdth; 
float* KZWeXiYFmdpQdsgidKeG = MALLOC_CALL(sizeof(float)*cRtIUoZRPICuQEOZOSzT); 
MWCNNLayer::call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), cRtIUoZRPICuQEOZOSzT, 
QMgBqCuvjnbWHWiVPEwn, pdleXafalaHAmketaFyq); CUDA_CALL(hipMemcpy(olKGEIcsxmLSoMhRhEtP, 
KZWeXiYFmdpQdsgidKeG, sizeof(float)*cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); 
free(KZWeXiYFmdpQdsgidKeG); fclose(QMgBqCuvjnbWHWiVPEwn);  } void 
MWElementwiseAffineLayerImpl::loadOffset(const char* gsJtSpgIkTNvahoTFqow) { 
FILE* QMgBqCuvjnbWHWiVPEwn = MWCNNLayer::openBinaryFile(gsJtSpgIkTNvahoTFqow); 
assert(QMgBqCuvjnbWHWiVPEwn); long int cRtIUoZRPICuQEOZOSzT = 
gTcJMwtYuwiqqUmqvKhT*gcGbhKACQPAogUYXHedj*gNROjwaqhxDPvBWUCUcQ; float* 
KZWeXiYFmdpQdsgidKeG = MALLOC_CALL(sizeof(float)*cRtIUoZRPICuQEOZOSzT); 
MWCNNLayer::call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), cRtIUoZRPICuQEOZOSzT, 
QMgBqCuvjnbWHWiVPEwn, gsJtSpgIkTNvahoTFqow); CUDA_CALL(hipMemcpy(fYaOQTeunPwVjnhhTECh, 
KZWeXiYFmdpQdsgidKeG, sizeof(float)*cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); 
free(KZWeXiYFmdpQdsgidKeG); fclose(QMgBqCuvjnbWHWiVPEwn);  } } 