//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: DeepLearningNetwork.cu
//
// GPU Coder version                    : 2.2
// CUDA/C/C++ source code generated on  : 10-Dec-2021 14:26:27
//

// Include Files
#include "DeepLearningNetwork.h"
#include "myAlexNetGPU_internal_types.h"
#include "MWCNNLayer.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
#include "MWElementwiseAffineLayer.hpp"
#include "MWFCLayer.hpp"
#include "MWFusedConvActivationLayer.hpp"
#include "MWInputLayer.hpp"
#include "MWMaxPoolingLayer.hpp"
#include "MWNormLayer.hpp"
#include "MWOutputLayer.hpp"
#include "MWReLULayer.hpp"
#include "MWSoftmaxLayer.hpp"
#include "MWTensor.hpp"
#include "MWTensorBase.hpp"
#include <cstdio>
#include <cstdlib>

// Named Constants
const char *errorString{
    "Abnormal termination due to: %s.\nError in %s (line %d)."};

const char *errStringBase{
    "Error during execution of the generated code. %s at line: %d, file: "
    "%s\nExiting program execution ...\n"};

// Function Declarations
static void checkCleanupCudaError(hipError_t errCode, const char *file,
                                  unsigned int line);

static void checkRunTimeError(const char *errMsg, const char *file,
                              unsigned int line);

namespace coder {
static void DeepLearningNetwork_callDelete(alexnet0_0 *obj);

}

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void alexnet0_0::allocate()
{
  targetImpl->allocate(290400, 2);
  for (int idx{0}; idx < 19; idx++) {
    layers[idx]->allocate();
  }
  (static_cast<MWTensor<float> *>(inputTensors[0]))
      ->setData(layers[0]->getLayerOutput(0));
}

//
// Arguments    : void
// Return Type  : void
//
void alexnet0_0::cleanup()
{
  deallocate();
  for (int idx{0}; idx < 19; idx++) {
    layers[idx]->cleanup();
  }
  if (targetImpl) {
    targetImpl->cleanup();
  }
  isInitialized = false;
  checkCleanupCudaError(hipGetLastError(), __FILE__, __LINE__);
}

//
// Arguments    : void
// Return Type  : void
//
void alexnet0_0::deallocate()
{
  targetImpl->deallocate();
  for (int idx{0}; idx < 19; idx++) {
    layers[idx]->deallocate();
  }
}

//
// Arguments    : void
// Return Type  : void
//
void alexnet0_0::postsetup()
{
  targetImpl->postSetup(layers, numLayers);
}

//
// Arguments    : void
// Return Type  : void
//
void alexnet0_0::resetState()
{
}

//
// Arguments    : void
// Return Type  : void
//
void alexnet0_0::setSize()
{
  for (int idx{0}; idx < 19; idx++) {
    layers[idx]->propagateSize();
  }
  allocate();
  postsetup();
}

//
// Arguments    : void
// Return Type  : void
//
void alexnet0_0::setup()
{
  if (isInitialized) {
    resetState();
  } else {
    targetImpl->preSetup();
    targetImpl->setAutoTune(true);
    (static_cast<MWInputLayer *>(layers[0]))
        ->createInputLayer(targetImpl, inputTensors[0], "SSCB", 0);
    (static_cast<MWElementwiseAffineLayer *>(layers[1]))
        ->createElementwiseAffineLayer(
            targetImpl, layers[0]->getOutputTensor(0), 227, 227, 3, 227, 227, 3,
            false, 1, 1,
            "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_data_scale.bin",
            "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_data_offset.bin", "SSCB",
            0);
    (static_cast<MWFusedConvActivationLayer *>(layers[2]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[1]->getOutputTensor(0), 11, 11, 3, 96, 4, 4,
            0, 0, 0, 0, 1, 1, 1,
            "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_conv1_w.bin",
            "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_conv1_b.bin", 0.0,
            MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB", 1);
    (static_cast<MWNormLayer *>(layers[3]))
        ->createNormLayer(targetImpl, layers[2]->getOutputTensor(0), 5, 0.0001,
                          0.75, 1.0, "SSCB", 0);
    (static_cast<MWMaxPoolingLayer *>(layers[4]))
        ->createMaxPoolingLayer<float, float>(
            targetImpl, layers[3]->getOutputTensor(0), 3, 3, 2, 2, 0, 0, 0, 0,
            0, 0, "FLOAT", 1, "SSCB", 1);
    (static_cast<MWFusedConvActivationLayer *>(layers[5]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[4]->getOutputTensor(0), 5, 5, 48, 128, 1, 1,
            2, 2, 2, 2, 1, 1, 2,
            "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_conv2_w.bin",
            "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_conv2_b.bin", 0.0,
            MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB", 0);
    (static_cast<MWNormLayer *>(layers[6]))
        ->createNormLayer(targetImpl, layers[5]->getOutputTensor(0), 5, 0.0001,
                          0.75, 1.0, "SSCB", 1);
    (static_cast<MWMaxPoolingLayer *>(layers[7]))
        ->createMaxPoolingLayer<float, float>(
            targetImpl, layers[6]->getOutputTensor(0), 3, 3, 2, 2, 0, 0, 0, 0,
            0, 0, "FLOAT", 1, "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[8]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[7]->getOutputTensor(0), 3, 3, 256, 384, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_conv3_w.bin",
            "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_conv3_b.bin", 0.0,
            MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB", 1);
    (static_cast<MWFusedConvActivationLayer *>(layers[9]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[8]->getOutputTensor(0), 3, 3, 192, 192, 1, 1,
            1, 1, 1, 1, 1, 1, 2,
            "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_conv4_w.bin",
            "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_conv4_b.bin", 0.0,
            MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[10]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[9]->getOutputTensor(0), 3, 3, 192, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 2,
            "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_conv5_w.bin",
            "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_conv5_b.bin", 0.0,
            MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB", 1);
    (static_cast<MWMaxPoolingLayer *>(layers[11]))
        ->createMaxPoolingLayer<float, float>(
            targetImpl, layers[10]->getOutputTensor(0), 3, 3, 2, 2, 0, 0, 0, 0,
            0, 0, "FLOAT", 1, "SSCB", 0);
    (static_cast<MWFCLayer *>(layers[12]))
        ->createFCLayer(targetImpl, layers[11]->getOutputTensor(0), 9216, 4096,
                        "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_fc6_w.bin",
                        "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_fc6_b.bin",
                        "SSCB", 1);
    (static_cast<MWReLULayer *>(layers[13]))
        ->createReLULayer<float, float>(targetImpl,
                                        layers[12]->getOutputTensor(0), 0,
                                        "FLOAT", 1, "SSCB", 1);
    (static_cast<MWFCLayer *>(layers[14]))
        ->createFCLayer(targetImpl, layers[13]->getOutputTensor(0), 4096, 4096,
                        "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_fc7_w.bin",
                        "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_fc7_b.bin",
                        "SSCB", 0);
    (static_cast<MWReLULayer *>(layers[15]))
        ->createReLULayer<float, float>(targetImpl,
                                        layers[14]->getOutputTensor(0), 0,
                                        "FLOAT", 1, "SSCB", 0);
    (static_cast<MWFCLayer *>(layers[16]))
        ->createFCLayer(targetImpl, layers[15]->getOutputTensor(0), 4096, 1000,
                        "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_fc8_w.bin",
                        "./codegen/lib/myAlexNetGPU/cnn_alexnet0_0_fc8_b.bin",
                        "SSCB", 1);
    (static_cast<MWSoftmaxLayer *>(layers[17]))
        ->createSoftmaxLayer(targetImpl, layers[16]->getOutputTensor(0), "SSCB",
                             0);
    (static_cast<MWOutputLayer *>(layers[18]))
        ->createOutputLayer(targetImpl, layers[17]->getOutputTensor(0), "SSCB",
                            0);
    outputTensors[0] = layers[18]->getOutputTensor(0);
    setSize();
  }
  isInitialized = true;
}

//
// Arguments    : hipError_t errCode
//                const char *file
//                unsigned int line
// Return Type  : void
//
static void checkCleanupCudaError(hipError_t errCode, const char *file,
                                  unsigned int line)
{
  if ((errCode != hipSuccess) && (errCode != hipErrorDeinitialized)) {
    printf(errorString, hipGetErrorString(errCode), file, line);
  }
}

//
// Arguments    : const char *errMsg
//                const char *file
//                unsigned int line
// Return Type  : void
//
static void checkRunTimeError(const char *errMsg, const char *file,
                              unsigned int line)
{
  printf(errStringBase, errMsg, line, file);
  exit(EXIT_FAILURE);
}

//
// Arguments    : alexnet0_0 *obj
// Return Type  : void
//
namespace coder {
static void DeepLearningNetwork_callDelete(alexnet0_0 *obj)
{
  if (&obj->isInitialized) {
    obj->cleanup();
  }
}

//
// Arguments    : void
// Return Type  : ::alexnet0_0
//
} // namespace coder
alexnet0_0::alexnet0_0()
{
  numLayers = 19;
  isInitialized = false;
  targetImpl = 0;
  layers[0] = new MWInputLayer;
  layers[0]->setName("data");
  layers[1] = new MWElementwiseAffineLayer;
  layers[1]->setName("data_normalization");
  layers[1]->setInPlaceIndex(0, 0);
  layers[2] = new MWFusedConvActivationLayer;
  layers[2]->setName("conv1_relu1");
  layers[3] = new MWNormLayer;
  layers[3]->setName("norm1");
  layers[4] = new MWMaxPoolingLayer;
  layers[4]->setName("pool1");
  layers[5] = new MWFusedConvActivationLayer;
  layers[5]->setName("conv2_relu2");
  layers[6] = new MWNormLayer;
  layers[6]->setName("norm2");
  layers[7] = new MWMaxPoolingLayer;
  layers[7]->setName("pool2");
  layers[8] = new MWFusedConvActivationLayer;
  layers[8]->setName("conv3_relu3");
  layers[9] = new MWFusedConvActivationLayer;
  layers[9]->setName("conv4_relu4");
  layers[10] = new MWFusedConvActivationLayer;
  layers[10]->setName("conv5_relu5");
  layers[11] = new MWMaxPoolingLayer;
  layers[11]->setName("pool5");
  layers[12] = new MWFCLayer;
  layers[12]->setName("fc6");
  layers[13] = new MWReLULayer;
  layers[13]->setName("relu6");
  layers[13]->setInPlaceIndex(0, 0);
  layers[14] = new MWFCLayer;
  layers[14]->setName("fc7");
  layers[15] = new MWReLULayer;
  layers[15]->setName("relu7");
  layers[15]->setInPlaceIndex(0, 0);
  layers[16] = new MWFCLayer;
  layers[16]->setName("fc8");
  layers[17] = new MWSoftmaxLayer;
  layers[17]->setName("prob");
  layers[18] = new MWOutputLayer;
  layers[18]->setName("output");
  layers[18]->setInPlaceIndex(0, 0);
  targetImpl = new MWCudnnTarget::MWTargetNetworkImpl;
  inputTensors[0] = new MWTensor<float>;
  inputTensors[0]->setHeight(227);
  inputTensors[0]->setWidth(227);
  inputTensors[0]->setChannels(3);
  inputTensors[0]->setBatchSize(1);
  inputTensors[0]->setSequenceLength(1);
}

//
// Arguments    : void
// Return Type  : void
//
alexnet0_0::~alexnet0_0()
{
  try {
    if (isInitialized) {
      cleanup();
    }
    for (int idx{0}; idx < 19; idx++) {
      delete layers[idx];
    }
    if (targetImpl) {
      delete targetImpl;
    }
    delete inputTensors[0];
  } catch (...) {
  }
}

//
// Arguments    : void
// Return Type  : int
//
int alexnet0_0::getBatchSize()
{
  return inputTensors[0]->getBatchSize();
}

//
// Arguments    : int b_index
// Return Type  : float *
//
float *alexnet0_0::getInputDataPointer(int b_index)
{
  return (static_cast<MWTensor<float> *>(inputTensors[b_index]))->getData();
}

//
// Arguments    : void
// Return Type  : float *
//
float *alexnet0_0::getInputDataPointer()
{
  return (static_cast<MWTensor<float> *>(inputTensors[0]))->getData();
}

//
// Arguments    : int layerIndex
//                int portIndex
// Return Type  : float *
//
float *alexnet0_0::getLayerOutput(int layerIndex, int portIndex)
{
  return layers[layerIndex]->getLayerOutput(portIndex);
}

//
// Arguments    : int layerIndex
//                int portIndex
// Return Type  : int
//
int alexnet0_0::getLayerOutputSize(int layerIndex, int portIndex)
{
  return layers[layerIndex]->getOutputTensor(portIndex)->getNumElements() *
         sizeof(float);
}

//
// Arguments    : int b_index
// Return Type  : float *
//
float *alexnet0_0::getOutputDataPointer(int b_index)
{
  return (static_cast<MWTensor<float> *>(outputTensors[b_index]))->getData();
}

//
// Arguments    : void
// Return Type  : float *
//
float *alexnet0_0::getOutputDataPointer()
{
  return (static_cast<MWTensor<float> *>(outputTensors[0]))->getData();
}

//
// Arguments    : void
// Return Type  : void
//
void alexnet0_0::predict()
{
  for (int idx{0}; idx < 19; idx++) {
    layers[idx]->predict();
  }
}

//
// Arguments    : alexnet0_0 *obj
// Return Type  : void
//
namespace coder {
void DeepLearningNetwork_delete(alexnet0_0 *obj)
{
  DeepLearningNetwork_callDelete(obj);
}

//
// Arguments    : alexnet0_0 *obj
// Return Type  : void
//
void DeepLearningNetwork_setup(alexnet0_0 *obj)
{
  try {
    obj->setup();
  } catch (std::runtime_error const &err) {
    obj->cleanup();
    checkRunTimeError(err.what(), __FILE__, __LINE__);
  } catch (...) {
    obj->cleanup();
    checkRunTimeError("", __FILE__, __LINE__);
  }
}

} // namespace coder

//
// File trailer for DeepLearningNetwork.cu
//
// [EOF]
//
