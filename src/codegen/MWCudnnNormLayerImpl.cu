#include "MWCudnnNormLayerImpl.hpp"
#include "MWCudnnCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
 namespace MWCudnnTarget { MWNormLayerImpl::MWNormLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, unsigned FrpxvsDMwwgbpqHXWxmN,  double 
AFQBkxwYGKLsACiDKwRM,  double AHqhysOOIgbDpWZoPUFT,  double BNrGqqHwfmYKIqbDbnjx) : 
MWCNNLayerImpl(layer, ntwk_impl)  { 
CUDNN_CALL(hipdnnCreateLRNDescriptor(&cCXqPFPPcoHzYMDpnUxQ)); 
createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex()); 
CUDNN_CALL(hipdnnSetLRNDescriptor(cCXqPFPPcoHzYMDpnUxQ, 
FrpxvsDMwwgbpqHXWxmN, AFQBkxwYGKLsACiDKwRM, AHqhysOOIgbDpWZoPUFT, 
BNrGqqHwfmYKIqbDbnjx)); } MWNormLayerImpl::~MWNormLayerImpl() { } void 
MWNormLayerImpl::propagateSize() { MWTensorBase* opTensor = 
getLayer()->getOutputTensor(0); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor));  } void 
MWNormLayerImpl::predict() { MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor();  MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
hipdnnTensorDescriptor_t ipDesc = 
MWCNNLayerImpl::getCuDNNDescriptor(ipTensorBase); 
CUDNN_CALL(hipdnnLRNCrossChannelForward(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
cCXqPFPPcoHzYMDpnUxQ, HIPDNN_LRN_CROSS_CHANNEL, getOnePtr(), ipDesc, 
ipTensor->getData(), getZeroPtr(), *desc, opTensor->getData())); } void 
MWNormLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyLRNDescriptor(cCXqPFPPcoHzYMDpnUxQ)); } } 